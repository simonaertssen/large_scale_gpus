#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../DIEKUHDA/kuhda.h"
#include "omp.h"

#define NUMTHREADS 4
#define NUMTHREADSBUFF 16

/*
This script contains the same functionality as AllDeviceMultiplication2 but with a full buffer
run with
nvcc -O3 -Xcompiler -fopenmp -lcublas ../DIEKUHDA/kuhda.cu AllDeviceMultiplication3.cu && ./a.out 1000 500
*/

void TileHostToGPUBuff(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *h_matrix, matrix *d_tile, hipStream_t stream, matrix *memacc );
void TileGPUAddToHostBuff(unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *d_tile, matrix *h_matrix, hipStream_t stream, matrix *memacc );


int main(int argc, char* argv[]) {
    // prepare timer:
    double start = omp_get_wtime(), end; 

    // set matrix size
    unsigned int n = 5000;
    if (argc > 1){
        n = (unsigned int)atoi(argv[1]);
        if (n > 40960 ) {
            printf("matrix dimension too large..\n");
            return -1;
        }
    }
    printf("Matrix dimension = %lu.. \n", n);
    unsigned int m = n, k = n;

    // set tile size
    unsigned int x = n/2;
    if (argc > 2){
        x = (unsigned int)atoi(argv[2]);
        if (x > n ) {
            x = n/2;
            printf("Block size too large, setting block size to %d..\n", x);
        }
    }
    printf("Block size = %lu..\n", x);

	// Containers for host and device matrices
	matrix *h_A = kuhdaMallocM1(n, n); // diagonal A matrix
	matrix *h_B = kuhdaMallocM1(n, n); // diagonal B matrix
	matrix *h_C = kuhdaMallocM(n, n); // empty C matrix

    int abc, ABC = 3; // counters to loop through matrices
    int device, devicecount = 4;
    int stream, streamsperdevice = (int) pow(2, (int) n/x);

    /* The number of streams can be computed as:
    n/x = 1:  1 streams per device, 1 loop    2**1 = 2
    n/x = 2:  2 streams per device, 8 loops   2**2 = 4
    n/x = 3:  7 streams per device, 27 loops  2**3 = 8
    n/x = 4: 16 streams per device, 64 loops  2**4 = 16
    n/x = 5: 32 streams per device, 125 loops 2**5 = 32
    Take a maximum of 32.
    */
    streamsperdevice = streamsperdevice > 32 ? 32 : streamsperdevice;

    // parallel device warmup
    #pragma omp parallel for private(device) num_threads(devicecount)
    for (device = 0; device < devicecount; device ++) kuhdaWarmupDevice(device);
    
    GPUCHECK(hipGetDeviceCount(&devicecount));
    matrix *d_All[devicecount][ABC];

    int streamcount = streamsperdevice*devicecount;
    hipStream_t d_streams[streamcount];
    hipblasHandle_t handles[devicecount];
    matrix *membuffs[devicecount];

    MatMulTimer timer;

    // Check dimensions with regards to the available memory:
    x = kuhdaAdjustTileSizeForAvailableMemory(devicecount, n, x);

    printf("Allocating tiles A, B and C on %d devices..\n", devicecount);
    #pragma omp parallel for private(device, abc, stream) num_threads(NUMTHREADS)
    // Creat all dependencies:
    for (device = 0; device < devicecount; device++){
        GPUCHECK(hipSetDevice(device));
        CUBLASCHECK(hipblasCreate(&handles[device])); 

        membuffs[device] = kuhdaMallocMP(x, x);

        // GPUCHECK(hipHostAlloc(&membuffs[device][0], x*x*sizeof(double), hipHostMallocPortable));
        // GPUCHECK(hipHostAlloc(&membuffs[device][1], x*x*sizeof(double), hipHostMallocPortable));

        for (abc = 0; abc < ABC; ++abc){
            d_All[device][abc] = kuhdaMallocDeviceM(x, x);
        }
        
        for (stream = 0; stream < streamsperdevice; ++stream){
            GPUCHECK(hipStreamCreate(&d_streams[device + stream*devicecount]));
        }
    }

    printf("Computation start..\n");
    timer.Start();

    int streamindex = 0, currentdevice = 0, loopindex = 0;
    int mtile = 0, ntile = 0, ktile = 0;
    // Loop over rows of A:
    //#pragma omp parallel for private(mtile)
    for (mtile = 0; mtile < m/x; ++mtile){
        // Loop over columns of B:
        for (ntile = 0; ntile < n/x; ++ntile){
            // #pragma omp parallel for private(ktile) num_threads(NUMTHREADS)
            // Loop over columns of A and rows of B:
            for (ktile = 0; ktile < k/x; ++ktile){
                // Set device by using integer division: 0, 0, 0, 1, 1, 1, ...
                //currentdevice = streamindex/streamsperdevice;
                GPUCHECK(hipSetDevice(currentdevice));

                TileHostToGPUBuff(mtile*x, (mtile+1)*x, ktile*x, (ktile+1)*x, h_A, d_All[currentdevice][0], d_streams[streamindex], membuffs[currentdevice]); // Tile A
                TileHostToGPUBuff(ktile*x, (ktile+1)*x, ntile*x, (ntile+1)*x, h_B, d_All[currentdevice][1], d_streams[streamindex], membuffs[currentdevice]); // Tile B

                // We are using two different streams to try out
                GPUCHECK(hipStreamSynchronize(d_streams[streamindex]));
                GPUCHECK(hipStreamSynchronize(d_streams[streamindex]));

                // damn man dads not sooo fast.. yet
                kuhdamm(d_All[currentdevice][0], d_All[currentdevice][1], d_All[currentdevice][2], d_streams[streamindex], handles[currentdevice]);

                // kuhdaPrintDeviceM(d_All[currentdevice][2]);

                // Get the tile back
                TileGPUAddToHostBuff(mtile*x, (mtile+1)*x, ntile*x, (ntile+1)*x, d_All[currentdevice][2], h_C, d_streams[streamindex], membuffs[currentdevice]);

                currentdevice++;
                if (currentdevice != 0 && currentdevice%devicecount == 0) loopindex++;
                currentdevice = currentdevice%devicecount;
                streamindex = currentdevice + loopindex*devicecount;
                streamindex = streamindex%streamcount;

                // kuhdaPrintM(h_C);
            }
        }
    }


    timer.Stop();
    double timingResult = timer.GFLOPS_DGEMM(m, n, k);
    printf("GFLOPS = %.0lf\n", timingResult);

    //h_C->data[100] = 578.0;
    // Test the result for mistakes
	kuhdaTestM(0, n, 0, n, h_C);
    //printf("%lf  %lf \n%lf  %lf \n", h_C->data[(n-1)*x-1], h_C->data[(n-1)*x], h_C->data[n*x-1], h_C->data[n*x]);

    // Free all
    printf("Cleaning up ..\n");
    GPUCHECK(hipSetDevice(0));

	kuhdaFreeM(h_A, 'k');
	kuhdaFreeM(h_B, 'k');
    kuhdaFreeM(h_C, 'k');

    timer.Release();

    #pragma omp parallel for private(device, abc, stream) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
        GPUCHECK(hipSetDevice(device));
        CUBLASCHECK(hipblasDestroy(handles[device]));

        kuhdaFreeM(membuffs[device], 'p');
        // GPUCHECK(hipHostFree(membuffs[device][0]));
        // GPUCHECK(hipHostFree(membuffs[device][1]));

        for (abc = 0; abc < ABC; ++abc){
            kuhdaFreeM(d_All[device][abc], 'c');
        }

        for (stream = 0; stream < streamsperdevice; ++stream){
            GPUCHECK(hipStreamDestroy(d_streams[device + stream*devicecount]));
        }
        // Takes NO arguments
        GPUCHECK(hipDeviceReset());
    }

    end = omp_get_wtime(); 
    printf("Script took %.1f seconds \n", end - start);
	return 0;
}


void TileHostToGPUBuff(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *h_matrix, matrix *d_tile, hipStream_t stream, matrix *memacc )
{	
    // check input
    if (h_matrix == NULL || d_tile == NULL) INPUT_NULL_ERR;
    if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
    if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
    if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
    if (stream == NULL) INPUT_NULL_ERR;

    unsigned long cols = colstop - colstart, rows = rowstop - rowstart, i, j;

    #pragma omp parallel for private(i) num_threads(NUMTHREADSBUFF)
    for (i=rowstart; i<rowstop; ++i){
        for (j=colstart; j<colstop; ++j){
            // fill memacc with host-matrix data one (tile-)row at a time:
            // memacc[j-colstart] = h_matrix->data[i * h_matrix->c + j];
            memacc->data[(i - rowstart) * memacc->c + (j - colstart)] = h_matrix->data[i * h_matrix->c + j];
        }
    }
    
    GPUCHECK(hipMemcpyAsync((void*)&d_tile->data[0], (void*)&memacc->data[0], rows*cols*sizeof(double), hipMemcpyHostToDevice, stream));
}

void TileGPUAddToHostBuff( unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *d_tile, matrix *h_matrix, hipStream_t stream, matrix *memacc )
{
    // check input
    if (h_matrix == NULL || d_tile == NULL) INPUT_NULL_ERR;
    if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
    if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
    if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
    if (stream == NULL) INPUT_NULL_ERR;

    unsigned long cols = colstop - colstart, rows = rowstop - rowstart, i, j;

    GPUCHECK(hipMemcpyAsync((void*)&memacc->data[0], (void*)&d_tile->data[0], rows*cols*sizeof(double), hipMemcpyDeviceToHost, stream));
    GPUCHECK(hipStreamSynchronize(stream));

    #pragma omp parallel for private(i) num_threads(NUMTHREADSBUFF)
    for (i = rowstart; i < rowstop; ++i){
        for (j = colstart; j < colstop; ++j){
            h_matrix->data[i * h_matrix->c + j] += memacc->data[(i - rowstart) * memacc->c + (j - colstart)];
        }
    }
}
