
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include "math.h"
#include "hipblas.h"
#include <limits.h>

// This script contains some numerical tests to get to know cublas 
// and how to split up matrices in blocks for gpu computation.
// Run with:
// nvcc -lcublas -lgomp experiment_1.cu && ./a.out

void test(double * matrix, int dim){
    int i, j;
    long long sum = 0;
    int count = 0;
    for (i = 0; i < dim; ++i){
        for (j = 0; j < dim; ++j){
            sum += (long long) matrix[i*dim + j];
            count++;
        }
    }
    long long testdim = (long long)dim*(long long)dim*(long long)dim;
    if (testdim == sum){
        // This should be true because all matrices are filled with ones.
        printf("Test: count is %d, dim**3 is %d, sum is %.1ld \n", count, (long int)dim*(long int)dim*(long int)dim, sum);
    }
    }


int main(){
    // Counters and timers:
    int i, j, left, right;
    double t1, t2, gflops;

    // Size of testmatrix = pow(2,14) = 16384
    int n = 16384, sub_n = 25;
    int n_squared = n*n, sub_n_squared = sub_n*sub_n;

    // Allocate a matrix A and C of size n*n
    double *A = (double*) malloc(n_squared * sizeof(double));
    double *C = NULL;
    //cudaMalloc(&C, n_squared * sizeof(double))
    if (A == NULL) printf("A is NULL\n");
    
    if (hipMalloc(&C, n_squared * sizeof(double)) != 0){
        fprintf(stderr, "CudaMalloc failed: matrix is of size %ldGB which is larger than 16GB (V100 memory).\n", n_squared * sizeof(double) / 10e9);
        exit(-1);
        }
    
    // Fill as ones
    for (i = 0; i < n; i++){
        for (j = 0; j  < n; j++){
            *(A + i*n + j) = 1.0;
        }
    }

    // Test if correct:
    //test(A, n);

    // Now the CUDA part:
    // Create cublas instance and stream
    hipblasHandle_t handle;
    if ( hipblasCreate(&handle) != 0 ) printf("hipblasCreate faileds\n");
    hipStream_t *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
    if ( hipStreamCreate(&stream[0]) != 0 ) printf("hipStreamCreate faileds\n");

    // Set matrix coefficients
    double alpha = 1.0;
    double beta  = 0.0;

    // Set CUDA stream
    hipblasSetStream(handle, stream[0]);
    if ( hipblasSetStream(handle, stream[0]) != 0 ) printf("hipblasSetStream failed\n");
 
    // DGEMM: A = alpha*A*A + beta*A
    t1 = omp_get_wtime();
    
    #pragma parallel for
    for (int i = 0; i < 4; ++i){
        left = sub_n*i, right = sub_n*(i+1);

        hipSetDevice(i);

        // Send matrix to GPU:
        if ( hipblasSetMatrix(n, n, sizeof(double), A, n, C, n) != 0 ) printf("hipblasSetMatrix faileds\n");
    
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, C, n, C, n, &beta, C, n);

        // Return the matrix
        if ( hipblasGetMatrix(n, n, sizeof(double*), C, n, A, n) != 0 ) printf("hipblasGetMatrix faileds\n");

    }
    
    //cudaDeviceSynchronize();
    t2 = omp_get_wtime();

    // Time the computations as in "How to compute GFLOPS for GEMM BLAS?" - nvidia forum
    // See: https://devtalk.nvidia.com/default/topic/482834/how-to-compute-gflops-for-gemm-blas/
    printf("Elapsed itme is %lf seconds\n", t2 - t1);
    gflops = 4*(long long)(n_squared * (2*n + 2)) / (10e9 *(t2 - t1));
    printf("Timed %lf GFLPS .. hah \n", gflops);

    test(A,n);

    free(A);
    hipFree(C);
    hipblasDestroy(handle);

    // https://github.com/zchee/cuda-sample/blob/master/0_Simple/matrixMulCUBLAS/matrixMulCUBLAS.cpp
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits.
    hipDeviceReset();

    return 0;
}