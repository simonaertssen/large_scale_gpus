
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include "math.h"
#include "hipblas.h"
#include <limits.h>

// PROBLEMS WITH SIZE OF MATRIX A, DUE TO INDICES I AND J
// Results from the latest session:
// n = 2*16384
// Time = 9.0686 seconds
// 775.978 GFLOPS


// This script contains some numerical tests to get to know cublas 
// and how to split up matrices in blocks for gpu computation.
// Run with:
// nvcc -lcublas -lgomp experiment_1.cu && ./a.out

void test(double * matrix, int dim){
    unsigned int i, j;
    double sum = 0.0;
    int count = 0;
    for (i = 0; i  < dim; ++i){
        for (j = 0; j  < dim; ++j){
            sum += matrix[i*dim + j];
            printf("Test: sum is %lf \n", sum);
            count++;
        }
    }
    printf("Test: count is %d, dim is %d, sum is %lf \n", count, dim*dim*dim, sum);
    }


int main(){
    // After running into trouble with large numbers, we need to check the max size
    // of the matrix dimensions.
    // printf("Max value of type int  = %d \n", INT_MAX);
    // printf("Max value of type uint = %zu \n", UINT_MAX);

    // Counters and timers:
    int i, j;
    double t1, t2, gflops;

    // Size of testmatrix = pow(2,14) = 16384
    int n = 16384*4;
    printf("n = %d\n", n);

    // Allocate a matrix A and C of size n*n
    double *A = (double*) malloc(n * n * sizeof(double));
    double *C = NULL;
    //cudaMalloc(&C, n * n * sizeof(double))
    if (A == NULL) printf("A is NULL\n");
    if (hipMalloc(&C, n * n * sizeof(double)) != 0) printf("CudaMalloc failed\n");
    
    // Fill as ones
    for (i = 0; i  < n; ++i){
        for (j = 0; j  < n; ++j){
            //A[i*n + j] = 1.0;
            printf("i = %d, j = %d\n", i, j);
            *(A + i*n + j) = 1.0;
        }
    }
    printf("i = %zu, j = %zu\n", i, j);

    // Test if correct:
    //test(A, n);

    // Now the CUDA part:
    // Create cublas instance and stream
    hipblasHandle_t handle;
    //cublasCreate(&handle)
    if ( hipblasCreate(&handle) != 0 ) printf("hipblasCreate faileds\n");


    hipStream_t *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
    //cudaStreamCreate(&stream[0]);
    if ( hipStreamCreate(&stream[0]) != 0 ) printf("hipStreamCreate faileds\n");

    // Send matrix to GPU:
    //cublasSetMatrix(n, n, sizeof(double), A, n, C, n);
    if ( hipblasSetMatrix(n, n, sizeof(double), A, n, C, n) != 0 ) printf("hipblasSetMatrix faileds\n");

    // Set matrix coefficients
    double alpha = 1.0;
    double beta  = 0.0;

    // Set CUDA stream
    //cublasSetStream(handle, stream[0]);
    if ( hipblasSetStream(handle, stream[0]) != 0 ) printf("hipblasSetStream faileds\n");

 
    // DGEMM: A = alpha*A*A + beta*A
    t1 = omp_get_wtime();
    hipblasDgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, n, n,
                    &alpha,
                    C, n,
                    C, n,
                    &beta,
                    C, n);

    hipDeviceSynchronize();
    t2 = omp_get_wtime();

    // Time the computations as in "How to compute GFLOPS for GEMM BLAS?" - nvidia forum
    printf("Time: %lf \n", t2-t1);
    gflops = ((long int)n*(long int)n*(2*(long int)n + 2)) / (10e9 *(t2 - t1));
    //gflops = (2 * (long int)n * (long int)n * (long int)n ) / (10e9 *(t2 - t1));

    //printf("Compute: (%d*%d*(2*%d + 2)) / (10e9 *(%lf - %lf)) \n", n, n, n, t2, t1);
    //printf("Compute: %d \n", n*n*n);
    printf("Timed %lf GFLPS .. hah \n", gflops);

    //cublasGetMatrix(n, n, sizeof(double*), C, n, A, n);
    if ( hipblasGetMatrix(n, n, sizeof(double*), C, n, A, n) != 0 ) printf("hipblasGetMatrix faileds\n");

    
    // test(A,n);

    free(A);
    hipFree(C);
    hipblasDestroy(handle);

    // https://github.com/zchee/cuda-sample/blob/master/0_Simple/matrixMulCUBLAS/matrixMulCUBLAS.cpp
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits.
    hipDeviceReset();

    return 0;
}