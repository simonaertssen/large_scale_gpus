#include <stdio.h>
#include "../DIEKUHDA/kuhda.h"
#include "omp.h"
#include <assert.h>

// run with 
// nvcc -O3 -Xcompiler -fopenmp -lcublas ../DIEKUHDA/kuhda.c OMPspeedTest.c && ./a.out

/*
Here we will investigate which OMP approach is the fastest when performing loops in parallel
*/

#define NUMTHREADS 4


int main() {

  int n = 10, tiledim = n/2, tilesize = tiledim*tiledim*sizeof(double);
  int device, devicecount = 4;
  int i, j;

  Timer timer;
  timer.Start();
  kuhdaWarmup(4);
  float elapsedtime = timer.Stop();
  printf("Warmup took %f ms\n", elapsedtime);

  // Containers for host and device matrices
  matrix *h_A = kuhdaMallocMP1(n, n), *d_A[devicecount];
  //double *test = (double*) malloc(tilesize);
  double *hostbuffer[devicecount], *hostbuffer_singlerow[devicecount];

  int streamsperdevice = 2;
  int stream, streamcount = streamsperdevice*devicecount;
  hipStream_t d_streams[streamcount];

  /*
  // Time the allocation loop
  printf("Timing the (de)allocation loops\n");
  timer.Start();
  for (device = 0; device < devicecount; device++){
      printf("Using %d OMP threads\n", (int)omp_get_num_threads());
      GPUCHECK(hipSetDevice(device));
      d_A[device] = kuhdaMallocDeviceM(tiledim, tiledim);
      GPUCHECK(hipHostMalloc((void**) &hostbuffer[device], tilesize));
      GPUCHECK(hipHostMalloc((void**) &hostbuffer_singlerow[device], tiledim*sizeof(double)));

      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamCreate(&d_streams[stream + streamsperdevice*device]));
      }
  }
  elapsedtime = timer.Stop();
  printf("Simple allocation took %f ms\n", elapsedtime);

  timer.Start();
  kuhdaFreeM(h_A, 'p');
  for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      kuhdaFreeM(d_A[device], 'c');
      hipFree(hostbuffer[device]);
      hipFree(hostbuffer_singlerow[device]);

      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamDestroy(d_streams[stream + streamsperdevice*device]));
      }
      GPUCHECK(hipDeviceReset());
  }
  elapsedtime = timer.Stop();
  printf("Simple destruction took %f ms\n", elapsedtime);

  
  // Time the allocation loop
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      printf("Using %d OMP threads", (int)omp_get_num_threads());
      GPUCHECK(hipSetDevice(device));
      d_A[device] = kuhdaMallocDeviceM(tiledim, tiledim);
      GPUCHECK(hipHostMalloc((void**) &hostbuffer[device], tilesize));
      GPUCHECK(hipHostMalloc((void**) &hostbuffer_singlerow[device], tiledim*sizeof(double)));

      #pragma unroll
      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamCreate(&d_streams[stream + streamsperdevice*device]));
      }
  }
  elapsedtime = timer.Stop();
  printf("Parallel allocation took %f ms\n", elapsedtime);

  timer.Start();
  kuhdaFreeM(h_A, 'p');
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      kuhdaFreeM(d_A[device], 'c');
      hipFree(hostbuffer[device]);
      hipFree(hostbuffer_singlerow[device]);

      #pragma unroll
      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamDestroy(d_streams[stream + streamsperdevice*device]));
      }
      GPUCHECK(hipDeviceReset());
  }
  elapsedtime = timer.Stop();
  printf("Parallel destruction took %f ms\n", elapsedtime);

 */

  // Time the tiling operation:
  printf("Timing the tiling operations\n");
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      printf("Using %d OMP threads\n", (int)omp_get_num_threads());
      GPUCHECK(hipSetDevice(device));
      d_A[device] = kuhdaMallocDeviceM(tiledim, tiledim);
      GPUCHECK(hipHostMalloc((void**) &hostbuffer[device], tilesize));
      GPUCHECK(hipHostMalloc((void**) &hostbuffer_singlerow[device], tiledim*sizeof(double)));

      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamCreate(&d_streams[stream + streamsperdevice*device]));
      }
  }

  kuhdaFillWithValue(h_A, 1.0);
  timer.Start();
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    TileHostToGPU(0, tiledim, 0, tiledim, h_A, d_A[device], d_streams[device*streamsperdevice]);
  }
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  elapsedtime = timer.Stop();
  printf("Simple tile transfer H2D took %f ms\n", elapsedtime);

  kuhdaFillWithValue(h_A, 0.0);

  timer.Start();
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    TileGPUToHost(0, tiledim, 0, tiledim, d_A[device], h_A, d_streams[device*streamsperdevice]);
  }
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  elapsedtime = timer.Stop();
  printf("Simple tile transfer D2H took %f ms\n", elapsedtime);
  kuhdaTestForValue(h_A, 1.0);

  kuhdaFillWithValue(h_A, 2.0);
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    TileHostToGPU(0, tiledim, 0, tiledim, h_A, d_A[device], d_streams[device*streamsperdevice]);
  }
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  elapsedtime = timer.Stop();
  printf("Parallel tile transfer H2D took %f ms\n", elapsedtime);

  kuhdaFillWithValue(h_A, 0.0);

  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    TileHostToGPU(0, tiledim, 0, tiledim, h_A, d_A[device], d_streams[device*streamsperdevice]);
  }
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  elapsedtime = timer.Stop();
  printf("Parallel tile transfer D2H took %f ms\n", elapsedtime);
  kuhdaTestForValue(h_A, 2.0);

  kuhdaFillWithValue(h_A, -5.0);
  timer.Start();
  //#pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    for (i = 0; i < tiledim; ++i){
      for (j = 0; j < tiledim; ++j) hostbuffer_singlerow[device][j] = h_A->data[i * h_A->c + j];
      GPUCHECK(hipMemcpyAsync((void*)(&d_A[device]->data[0] + tiledim * i), (void*)hostbuffer_singlerow[device], tiledim*sizeof(double), hipMemcpyHostToDevice, d_streams[device*streamsperdevice + (int)(i%streamsperdevice)]));
    }
  }

  //#pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 0]));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 1]));
  }
  elapsedtime = timer.Stop();
  printf("Fast parallel tile transfer H2D took %f ms\n", elapsedtime);

  kuhdaFillWithValue(h_A, 0.0);

  timer.Start();
  //#pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    for (i = 0; i < tiledim; ++i){
      GPUCHECK(hipMemcpyAsync((void*)hostbuffer_singlerow[device], (void*)(&d_A[device]->data[0] + tiledim * i), tiledim*sizeof(double), hipMemcpyDeviceToHost, d_streams[device*streamsperdevice + (int)(i%streamsperdevice)]));
      for (j = 0; j < tiledim; ++j) h_A->data[i * h_A->c + j] += hostbuffer_singlerow[device][j];
    }
  }

  //#pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 0]));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 1]));
  }
  elapsedtime = timer.Stop();
  printf("Fast parallel tile transfer D2H took %f ms\n", elapsedtime);
  //kuhdaTestForValue(h_A, -5.0);


  printf("Final deallocation\n");
  kuhdaFreeM(h_A, 'p');
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      kuhdaFreeM(d_A[device], 'c');
      hipHostFree(hostbuffer[device]);
      hipHostFree(hostbuffer_singlerow[device]);

      #pragma unroll
      for (stream = 0; stream < streamsperdevice; ++stream){
        GPUCHECK(hipStreamDestroy(d_streams[stream + streamsperdevice*device]));
      }
      GPUCHECK(hipDeviceSynchronize());
      //GPUCHECK(hipDeviceReset());
  }

  return 0;
}
