#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../DIEKUHDA/kuhda.h"
#include "omp.h"
#include <assert.h>

/*
With this script we will be timing the performance of different approaches to memcopies between devices and the host.
These tests are to find which approach is the fastest to be used in our matrix multiplication algorithms.
*/

/*
__global__ void fillMatrix(matrix *A);
void checkMatrixIsOnes(matrix *A);
void checkMatrixIsRows(matrix *A);
*/

#define NUMTHREADS 4


int main() {

  int n = 10000, tiledim = n/2, tilesize = tiledim*tiledim*sizeof(double);
  int device, devicecount = 4;
  int verbose = 0, rep, reps = 5;
  int i, j;

  printf("Timing the tiling operations for average of %d reps for matrix size n = %d\n", reps, n);

  Timer timer;
  timer.Start();
  kuhdaWarmup(4);
  float elapsedtime = timer.Stop(), results = 0.f;
  printf("Warmup took %f ms\n", elapsedtime);

  // These numbers demarcate the limits of the tiles on the host matrix. For simplicity we are using 4 tiles.
  int destinations[4][4] = {{0, tiledim, 0, tiledim}, {0, tiledim, tiledim, n}, {tiledim, n, 0, tiledim}, {tiledim, n, tiledim, n}};

  // Containers for host and device matrices
	matrix *h_A, *d_A[devicecount];
  double *hostbuffer[devicecount];
  double *hostbuffer_singlerow[devicecount];

  int streamsperdevice = 2;
  int stream, streamcount = streamsperdevice*devicecount;
  hipStream_t d_streams[streamcount];

  // Time the allocation loop
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      h_A = kuhdaMallocM1(n, n);
      d_A[device] = kuhdaMallocDeviceM(tiledim, tiledim);
      GPUCHECK(hipHostMalloc((void**) &hostbuffer[device], tilesize));
      GPUCHECK(hipHostMalloc((void**) &hostbuffer_singlerow[device], tiledim*sizeof(double)));

      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamCreate(&d_streams[stream + streamsperdevice*device]));
      }
      GPUCHECK(hipDeviceSynchronize());
  }
  elapsedtime = timer.Stop();
  printf("Allocation took %f ms\n", elapsedtime);

  /* 
  #################################################################################################################
  
  1. Naive approach: pinned buffer of same size as tiles on host and hipMemcpyAsync on one stream, back and forth

  #################################################################################################################
  */

  results = 0.f;
  kuhdaFillWithValue(h_A, 1.0);
  for (rep = 0; rep < reps; rep ++){
    timer.Start();
    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      for (i = 0; i < tiledim; ++i){
        for (j = 0; j < tiledim; ++j){
          hostbuffer[device][i * tiledim + j] = h_A->data[(i + destinations[device][0]) * h_A->c + (j + destinations[device][2])];
        }
      }
      GPUCHECK(hipMemcpyAsync((void*)(&d_A[device]->data[0]), hostbuffer[device], tilesize, hipMemcpyHostToDevice, d_streams[device*streamsperdevice]));
    }

    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
    }
    elapsedtime = timer.Stop();
    results += elapsedtime;
  }
  printf("Naive H2D took %f ms\n", results/reps);
  kuhdaFillWithValue(h_A, 0.0);

  results = 0.f;
  for (rep = 0; rep < reps; rep ++){
    timer.Start();
    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      GPUCHECK(hipMemcpyAsync(hostbuffer[device], (void*)(&d_A[device]->data[0]), tilesize, hipMemcpyDeviceToHost, d_streams[device*streamsperdevice]));
      for (i = 0; i < tiledim; ++i){
        for (j = 0; j < tiledim; ++j){
          h_A->data[(i + destinations[device][0]) * h_A->c + (j + destinations[device][2])] = hostbuffer[device][i * tiledim + j];
        }
      }
    }

    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
    }
    elapsedtime = timer.Stop();
    results += elapsedtime;
  }
  printf("Naive D2H took %f ms\n", results/reps);
  kuhdaTestForValue(h_A, 1.0, verbose);

  /* 
  #################################################################################################################
  
  2. TileHostToGPU and TileGPUAddToHost with a single stream. Under the hood pinned memory is used.

  #################################################################################################################
  */
  
  results = 0.f;
  kuhdaFillWithValue(h_A, 2.0);
  for (rep = 0; rep < reps; rep ++){
    timer.Start();
    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      TileHostToGPU(destinations[device][0], destinations[device][1],destinations[device][2], destinations[device][3], h_A, d_A[device], d_streams[device*streamsperdevice]);
    }

    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
    }
    elapsedtime = timer.Stop();
    results += elapsedtime;
  }
  printf("Tiled H2D took %f ms\n", results/reps);

  kuhdaFillWithValue(h_A, 0.0);

  results = 0.f;
  for (rep = 0; rep < reps; rep ++){
    timer.Start();
    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      TileGPUToHost(destinations[device][0], destinations[device][1],destinations[device][2], destinations[device][3], d_A[device], h_A, d_streams[device*streamsperdevice]);
    }

    #pragma omp parallel for private(device) num_threads(NUMTHREADS)
    for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
    }
    elapsedtime = timer.Stop();
    results += elapsedtime;
  }
  printf("Tiled D2H took %f ms\n", results/reps);

  kuhdaTestForValue(h_A, 2.0, verbose);


  // Time the destruction loop
  timer.Start();
  kuhdaFreeM(h_A, 'k');
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      kuhdaFreeM(d_A[device], 'c');
      hipFree(hostbuffer[device]);
      hipFree(hostbuffer_singlerow[device]);

      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamDestroy(d_streams[stream + streamsperdevice*device]));
      }
      GPUCHECK(hipDeviceSynchronize());
      //gpuErrchk(hipDeviceReset());
  }
  elapsedtime = timer.Stop();
  printf("Destruction took %f ms\n", elapsedtime);

  return 0;
}

/*
__global__ void fillMatrix(matrix *A) {
	const int row = blockIdx.y * blockDim.y + threadIdx.y, col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < A->r && col < A->c) A->data[row * A->c + col] = row;
}

void checkMatrixIsOnes(matrix *A) {
  int i, j;
  for (i = 0; i < tilesize; ++i) for (j = 0; j < tilesize; ++j) assert(A->data[i * A->c + j] == 1.0);
}

void checkMatrixIsRows(matrix *A) {
  int i, j;
  for (i = 0; i < tilesize; ++i) for (j = 0; j < tilesize; ++j) assert(A->data[i * A->c + j] == (double)i);
}
*/