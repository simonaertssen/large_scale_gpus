#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../DIEKUHDA/kuhda.h"
#include "omp.h"
#include <assert.h>

/*
With this script we will be timing the performance of different approaches to memcopies between devices and the host.
These tests are to find which approach is the fastest to be used in our matrix multiplication algorithms.
*/
#define GPUTHREADS 32

__global__ void fillMatrix(matrix *A);
void checkMatrixIsOnes(matrix *A);
void checkMatrixIsRows(matrix *A);



int main() {

  int n = 10000, tiledim = n/2, tilesize = tiledim*tiledim*sizeof(double);
  int device, devicecount = 4, NUMTHREADS = devicecount;
  int i, j;

  dim3 block(THREADS, THREADS);
  dim3 grid(ceil(((float)tiledim)/block.x), ceil(((float)tiledim)/block.y));

  Timer timer;
  timer.Start();
  kuhdaWarmup(4);
  float elapsedtime = timer.Stop();
  printf("Warmup took %f ms\n", elapsedtime);

  // Containers for host and device matrices
	matrix *h_A  = kuhdaMallocMP1(n, n);
	matrix *d_A[devicecount] = NULL;
  double *hostbuffer[devicecount] = NULL;
  double *hostbuffer_singlerow[devicecount] = NULL;

  int streamsperdevice = 2;
  int stream, streamcount = streamsperdevice*devicecount;
  hipStream_t d_streams[streamcount];

  // Time the allocation loop
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      printf("Using %d opm threads", (int)omp_get_num_threads());
      GPUCHECK(hipSetDevice(device));
      d_A[device] = kuhdaMallocDeviceM(tiledim, tiledim);
      GPUCHECK(hipHostMalloc((void)&hostbuffer[device], tilesize));
      GPUCHECK(hipHostMalloc((void)&hostbuffer_singlerow[device], tiledim*sizeof(double)));

      #pragma unroll
      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamCreate(&d_streams[stream + streamsperdevice*device]));
      }
  }
  elapsedtime = timer.Stop()
  printf("Allocation took %f ms\n", elapsedtime);


  // 1. Naive approach: pinned buffer on host and hipMemcpyAsync on one stream, back and forth
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    for (i = 0; i < tiledim; ++i){
  		for (j = 0; j < tiledim; ++j) hostbuffer[device][i * tiledim + j] = h_A->data[i * h_A->c + j];
    }
    GPUCHECK(hipMemcpyAsync((void*)(&d_A->data), (void*)hostbuffer[device], tilesize, hipMemcpyHostToDevice, d_streams[device*streamsperdevice]));
  }
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  timer.Stop();
  printf("Naive H2D took %f ms\n", elapsedtime);

  // Checking results and filling with new parameters
  checkMatrixIsOnes(d_A);
	fill_matrix<<<grid, block>>>(d_A);

  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    for (i = 0; i < tiledim; ++i){
  		for (j = 0; j < tiledim; ++j) d_A->data[i * d_A->c + j] = hostbuffer[device][i * tilesize + j];
    }
    GPUCHECK(hipMemcpyAsync((void*)hostbuffer[device], (void*)(&h_A->data), tilesize, hipMemcpyDeviceToHost, d_streams[device*streamsperdevice]));
  }
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  timer.Stop();
  printf("Naive D2H took %f ms\n", elapsedtime);

  checkMatrixIsRows(h_A);


  // 2. TileHostToGPU and TileGPUAddToHost with a single stream
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    TileHostToGPU(0, tiledim, 0, tiledim, h_A, d_A[device], d_streams[device*streamsperdevice]);
  }

  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  timer.Stop();
  printf("Tiled approach H2D took %f ms\n", elapsedtime);

  checkMatrixIsOnes(d_A);
	fill_matrix<<<grid, block>>>(d_A);

  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    TileGPUToHost(0, tiledim, 0, tiledim,d_A[device], h_A, d_streams[device*streamsperdevice]);
  }

  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice]));
  }
  timer.Stop();
  printf("Tiled approach D2H took %f ms\n", elapsedtime);

  checkMatrixIsRows(h_A);


  // 3. similar code to TileHostToGPU and TileGPUAddToHost but with special buffer and two streams
  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    for (i = 0; i < tiledim; ++i){
  		for (j = 0; j < tiledim; ++j) hostbuffer_singlerow[device][j] = h_A->data[i * h_A->c + j];
      GPUCHECK(hipMemcpyAsync((void*)(&d_A->data), (void*)hostbuffer_singlerow[device], tiledim*sizeof(double), hipMemcpyHostToDevice, d_streams[device*streamsperdevice + (int)(i%streamsperdevice)]));
    }
  }
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 0]));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 1]));
  }
  timer.Stop();
  printf("Fast line H2D took %f ms\n", elapsedtime);

  // Checking results and filling with new parameters
  checkMatrixIsOnes(d_A);
	fill_matrix<<<grid, block>>>(d_A);

  timer.Start();
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    for (i = 0; i < tiledim; ++i){
  		for (j = 0; j < tiledim; ++j) d_A->data[i * d_A->c + j] = hostbuffer_singlerow[device][j];
      GPUCHECK(hipMemcpyAsync((void*)hostbuffer_singlerow[device], (void*)(&h_A->data), tiledim*sizeof(double), hipMemcpyDeviceToHost, d_streams[device*streamsperdevice + (int)(i%streamsperdevice)]));
    }
  }
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
    GPUCHECK(hipSetDevice(device));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 0]));
    GPUCHECK(hipStreamSynchronize(d_streams[device*streamsperdevice + 1]));
  }
  timer.Stop();
  printf("Fast line D2H took %f ms\n", elapsedtime);

  checkMatrixIsRows(h_A);



  // Time the destruction loop
  timer.Start()
  kuhdaFreeM(h_A, 'p');
  #pragma omp parallel for private(device) num_threads(NUMTHREADS)
  for (device = 0; device < devicecount; device++){
      GPUCHECK(hipSetDevice(device));
      kuhdaFreeM(d_A[device], 'c');
      hipFree(hostbuffer[device]);
      hipFree(hostbuffer_singlerow[device]);

      #pragma unroll
      for (stream = 0; stream < streamsperdevice; ++stream){
          GPUCHECK(hipStreamDestroy(d_streams[stream + streamsperdevice*device]));
      }
      gpuErrchk(hipDeviceReset());
  }
  elapsedtime = timer.Stop()
  printf("Destruction took %f ms\n", elapsedtime);

  return 0
}


__global__ void fillMatrix(matrix *A) {
	const int row = blockIdx.y * blockDim.y + threadIdx.y, col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < A->r && col < A->c) A->data[row * A->c + col] = row;
}

void checkMatrixIsOnes(matrix *A) {
  int i, j;
  for (i = 0; i < tilesize; ++i) for (j = 0; j < tilesize; ++j) assert(A->data[i * A->c + j] == 1.0);
}

void checkMatrixIsRows(matrix *A) {
  int i, j;
  for (i = 0; i < tilesize; ++i) for (j = 0; j < tilesize; ++j) assert(A->data[i * A->c + j] == (double)i);
}
