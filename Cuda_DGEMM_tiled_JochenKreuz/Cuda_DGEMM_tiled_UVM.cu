// include libraries
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"

#define nstreams 1    

int main () {

  // banner
  printf ("\n\nGPU DGEMM Exercise\n");
  printf (    "==========================================\n");
  printf (  "\nTiled Matrix-Matrix Multiplication\n");
  printf (    "Using NVIDIA cuBLAS Library\n");

  // echo device data
  int idevice = 0;
  hipSetDevice(idevice);
  hipDeviceProp_t dprops;
  hipGetDeviceProperties( &dprops, idevice );
  printf ("\nDevice name = %s, with compute capability %d.%d \n", 
	  dprops.name, dprops.major, dprops.minor);

  // define parameters
  int n = 32768;   // matrix dimension - all matrices being multiplied will be square
  int m = 4096;    // tile size - tiles will be square, n must be divisible by m !!
  printf ("\nMatrix sizes: %d x %d, tile size: %d x %d\n", n,n,m,m);
  if ( ( n % m ) != 0  ) {
    printf ("\nmatrix size (n) has to be devisible by tile  size (m) !");
    return 0 ;
  }   
  printf ("Number of Streams: %d", nstreams);
  
  // allocate arrays
  double *a;
  double *b;
  double *c;
  a = (double *) malloc ( n*n*sizeof(double) );
  b = (double *) malloc ( n*n*sizeof(double) );
  c = (double *) malloc ( n*n*sizeof(double) );
  
  // initialize data
  #pragma omp parallel for
  for ( int row = 0; row<n; row++ ) {
    for ( int col = 0; col<n; col++ ) {
      // data in row-major format
      a[row*n+col] = row + col;
      b[row*n+col] = (row == col )  ? 1.0 : 0.0;
      c[row*n+col] = 0.0;
    }
  }

  // create communcations arrays
  double *pa;
  double *pb;
  double *pc;
  hipMallocManaged ( &pa, m*m*sizeof(double) );
  hipMallocManaged ( &pb, m*m*sizeof(double) );
  hipMallocManaged ( &pc, m*m*sizeof(double) );
	  
  // create a handle to cuBlas
  hipblasHandle_t cublasHandle;
  hipblasCreate( &cublasHandle );

  int ntiles = n/m;
  
  // record start time
  hipEvent_t t_start;
  hipEvent_t t_end;
  hipEvent_t compute_end;
  hipEventCreate (&t_start);
  hipEventCreate (&t_end);
  hipEventCreate (&compute_end);
  hipEventRecord (t_start,0);

  // caches for indices of previous tile to write back results 
  // from pinned buffer to original result matrix
  int prowtile;
  int pcoltile;

  // PERFORM MULTIPLICATION
  {

    double alpha = 1.0;
    double beta = 1.0; 

    int itile = 0;

    // loop over inner tile dimension
    for ( int iktile = 0; iktile < ntiles; iktile++ ) {
  
      // loop over row tiles
      for ( int irowtile = 0; irowtile < ntiles; irowtile++ ) {

        // loop over column tiles
        for ( int icoltile = 0; icoltile < ntiles; icoltile++ ) {
	  
	  if ( itile >= 1 ) {

	    hipEventSynchronize (compute_end); // needed since cublasDgemm call is asynchronous and copy is only done 
	                                        // on page fault (if results has already been written to pc
	    // copy result in pinned buffer back to global matrix
            # pragma omp parallel for
	    for ( int i=0; i<m; i++ ) {
	      for ( int j=0; j<m; j++ ) {
		c[(prowtile*m+i)*n+pcoltile*m+j] = pc[i*m+j];
	      }
	    }
	  } 

	  // copy next tile to pinned buffer
          # pragma omp parallel for
	  for ( int i = 0; i < m; i++ ) {
	    for ( int j = 0; j < m; j++ ) {
	      pa[i*m+j] = a[(irowtile*m+i)*n+iktile*m+j];
	      pb[i*m+j] = b[(iktile*m+i)*n+icoltile*m+j];
	      pc[i*m+j] = c[(irowtile*m+i)*n+icoltile*m+j];
	    }
	  }

	  // perform dgemm
	  hipblasDgemm ( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, m, m, &alpha, pa, m, pb, m, &beta, pc, m );
	  hipEventRecord (compute_end, 0);
	  prowtile = irowtile;
	  pcoltile = icoltile;

	  // go to next tile
	  itile++;

	}
      }
    }

    hipEventSynchronize (compute_end);	  // be sure that last cublasDgemm call has finished
    // copy result in pinned buffer back to source 
    # pragma omp parallel for
    for ( int i=0; i<m; i++ ) {
      for ( int j=0; j<m; j++ ) {
	c[(prowtile*m+i)*n+pcoltile*m+j] = pc[i*m+j];
      }
    }

  } // END OF PERFORM MULTIPLICATION



  // record end time
  hipEventRecord (t_end,0);
  hipEventSynchronize(t_end);
  float et;
  hipEventElapsedTime (&et, t_start, t_end);

  // check results
  printf ("\nchecking results: ");
  bool correct = true;
  int num_errors = 0;
  double abs_error, sum_abs_errors = 0;
# pragma omp parallel for
  for ( int row = 0;  row < n; row++ ) {
    for ( int col = 0; col < n; col++ ) {
      
      abs_error = fabs(c[row * n + col] - a[row * n + col] );
      sum_abs_errors += abs_error;
      if (  abs_error > 10e-1 ) {
	printf ("FAILED\n\nerror: c[%d]: %f != a[%d]: %f", 
		row * n + col,  c[row * n + col], row * n + col,  a[row * n + col]);
	correct = false;
	++num_errors;
	break;
      }
    }
  }
  
  // report results
  if ( correct ) {
    printf ("SUCCESS");
    printf ("\nSum abs errors: %f", sum_abs_errors);
    printf ("\nNumber of errors: %d", num_errors);
    printf("\nExecution time: %4.4f seconds\n", (double)et/1000.);     // cudaEventElapsedTime is in milliseconds
    printf(  "Gflop/s: %4.4f \n\n\n", 2.0e-6*n*n*n/et); // 2( * and + ) *n (inner dimension)*n^2(result size)/(time in ms.)
  } else {
    printf ("\nResult not correct (%d errors), check your code !\n", num_errors);
  }

  // clean up
  hipblasDestroy ( cublasHandle );
  hipEventDestroy ( t_start  );
  hipEventDestroy ( t_end );

  hipFree ( pa );
  hipFree ( pb );
  hipFree ( pc );

  free (a);
  free (b);
  free (c);

}
