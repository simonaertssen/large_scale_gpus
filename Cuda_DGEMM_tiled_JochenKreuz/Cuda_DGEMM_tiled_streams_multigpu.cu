// include libraries
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"

#define ngpus    4                   // Number of GPUs to use - cant be more than the # of gpus in the node
#define nstreams 3                   // Number of streams PER GPU


// Test for cuda errors
void cuerrmsg ( hipError_t cuerr, const char *msg) {
  if( cuerr ) {
    printf ("ERROR: %s\n", msg);
    abort();
  }
}

// Test for cuBLAS errors
void custatmsg ( hipblasStatus_t custat, const char *msg) {
  if( custat ) {
    printf ("ERROR: %s\n", msg);
    abort();
  }
}


int main () {

  // banner
  printf ("\n\nGPU Computing Advanced Workshop Exercise\n");
  printf (    "==========================================\n");
  printf (  "\nTiled Matrix-Matrix Multiplication\n");
  printf (    "Using cuBlas and streams on multiple GPUs \n");

  // define parameters
  int n = 32768;        // matrix dimension - all matrices being multiplied will be square
  int m = 4096;         // tile size - tiles will be square, n must be divisible by m
                        // on Juron: 40960, 8192

  printf ("\nMatrix sizes: %d x %d, tile size: %d x %d\n", n,n,m,m);
  if ( ( n % m ) != 0  ) {
    printf ("\nmatrix size (n) has to be devisible by tile  size (m) !");
    return 0 ;
  }   
  printf ("Number of GPUs: %d, Number of streams/GPU: %d", ngpus, nstreams);

  // echo device data
  for ( int idevice=0; idevice < ngpus; idevice++ ) {
    hipSetDevice(idevice);
    hipDeviceProp_t dprops;
    hipGetDeviceProperties( &dprops, idevice );
    printf ("\nDevice name = %s, with compute capability %d.%d \n", 
	    dprops.name, dprops.major, dprops.minor);
  }

  // allocate arrays for A, B and C matrices on the host
  double *a;
  double *b;
  double *c;
  a = (double *) malloc ( n*n*sizeof(double) );
  b = (double *) malloc ( n*n*sizeof(double) );
  c = (double *) malloc ( n*n*sizeof(double) );
  
  // initialize input data on host
#pragma omp parallel for
  for ( int row = 0; row<n; row++ ) {
    for ( int col = 0; col<n; col++ ) {
      // data in row-major format
      a[row*n+col] = row + col;
      b[row*n+col] = (row == col )  ? 1.0 : 0.0;
      c[row*n+col] = 0.0;
    }
  }
  
  
  // create pinned buffers for host<->device communcation
  hipError_t cuerr;
  double *pa;
  double *pb;
  double *pc;
  cuerr = hipHostMalloc ( &pa, m*m*sizeof(double)*nstreams*ngpus , hipHostMallocDefault);
  cuerrmsg ( cuerr, "hipHostMalloc pa");
  cuerr = hipHostMalloc ( &pb, m*m*sizeof(double)*nstreams*ngpus , hipHostMallocDefault);
  cuerrmsg ( cuerr, "hipHostMalloc pb");
  cuerr = hipHostMalloc ( &pc, m*m*sizeof(double)*nstreams*ngpus , hipHostMallocDefault);
  cuerrmsg ( cuerr, "hipHostMalloc pc");
	  
  // create a handle to cuBlas on each device
  hipblasStatus_t custat;
  hipblasHandle_t cublasHandle[ngpus]; 
  for ( int igpu=0; igpu<ngpus; igpu++ ) {
    cuerr = hipSetDevice(igpu);
    if ( cuerr ) {
      printf ("ERROR: hipSetDevice 1\n");
    }
    custat = hipblasCreate( &(cublasHandle[igpu]) );
    if ( custat ) {
      printf ("ERROR createing cuBlas context %d\n", igpu);
    }
    
  }

  // allocate space on device - 3 tiles for a, b, c
  double *d_a[ngpus];
  double *d_b[ngpus];
  double *d_c[ngpus];

  for ( int igpu=0; igpu<ngpus; igpu++ ) {
    cuerr = hipSetDevice(igpu);
    cuerrmsg ( cuerr, "hipSetDevice 2");
    cuerr = hipMalloc ( &(d_a[igpu]), nstreams*m*m*sizeof(double) );
    cuerrmsg ( cuerr, "hipMalloc d_a[]\n");
    cuerr = hipMalloc ( &(d_b[igpu]), nstreams*m*m*sizeof(double) );
    cuerrmsg ( cuerr, "hipMalloc d_b[]\n");
    cuerr = hipMalloc ( &(d_c[igpu]), nstreams*m*m*sizeof(double) );
    cuerrmsg ( cuerr, "hipMalloc d_c[]\n");
  }

  int offset = m*m;
  int ntiles = n/m;

  // create streams for each device
  hipStream_t myStreams[nstreams*ngpus];
  for ( int i=0; i<ngpus*nstreams; i++ ) {
    hipSetDevice(i/nstreams);
    cuerr = hipStreamCreate( &myStreams[i] );
    cuerrmsg ( cuerr, "hipStreamCreate \n");
  }

  // create events to signal when the D2H copy of result tiles has completed
  hipEvent_t bufferfilled[nstreams*ngpus];
  for ( int i=0; i<ngpus*nstreams; i++ ) {
    hipSetDevice (i/nstreams);
    hipEventCreate ( &bufferfilled[i] );
  }

  // record start time
  hipSetDevice(0);
  hipEvent_t t_start;
  hipEvent_t t_end;
  cuerr = hipEventCreate (&t_start);
  cuerrmsg ( cuerr, "hipEventCreate t_start \n");
  cuerr = hipEventCreate (&t_end);
  cuerrmsg ( cuerr, "hipEventCreate t_end \n");
  cuerr = hipEventRecord (t_start,0);
  cuerrmsg ( cuerr, "hipEventRecord \n");

  // caches for indices of previous tiles in streams
  int prowtile[nstreams*ngpus];
  int pcoltile[nstreams*ngpus];

  
  // PERFORM MULTIPLICATION
  {

    double alpha = 1.0;
    double beta = 1.0; 

    int ibuff = 0;
    int itile = 0;
    int igpu = 0;

    // loop over inner tile dimension
    for ( int iktile = 0; iktile < ntiles; iktile++ ) {
  
      // loop over row tiles
      for ( int irowtile = 0; irowtile < ntiles; irowtile++ ) {

        // loop over column tiles
        for ( int icoltile = 0; icoltile < ntiles; icoltile++ ) {

	  cuerr = hipSetDevice(igpu);
	  cuerrmsg ( cuerr, "hipSetDevice \n");

	  // first time accessing any device, don't need to empty result buffers 
	  if ( itile >= nstreams*ngpus ) {

	    // make sure that buffers are available
	    hipEventSynchronize ( bufferfilled[ibuff] );
	    cuerrmsg ( cuerr, "cudaEventSychronize \n" );
	    
	    // copy result in pinned buffer back to source 
            # pragma omp parallel for
	    for ( int i=0; i<m; i++ ) {
	      for ( int j=0; j<m; j++ ) {
		c[(prowtile[ibuff]*m+i)*n+pcoltile[ibuff]*m+j] = pc[ibuff*offset+i*m+j];
	      }
	    }
	  } 

	  // copy data to pinned buffer on host
          # pragma omp parallel for
	  for ( int i=0; i<m; i++ ) {
	    for ( int j=0; j<m; j++ ) {
	      pa[ibuff*offset+i*m+j] = a[(irowtile*m+i)*n+iktile*m+j];
	      pb[ibuff*offset+i*m+j] = b[(iktile*m+i)*n+icoltile*m+j];
	      pc[ibuff*offset+i*m+j] = c[(irowtile*m+i)*n+icoltile*m+j];
	    }
	  }

	  // copy input data to device
	  cuerr = hipMemcpyAsync ( &(d_a[igpu][(ibuff%nstreams)*offset]), &pa[ibuff*offset], m*m*sizeof(double), hipMemcpyHostToDevice, myStreams[ibuff] );
	  cuerrmsg ( cuerr, "hipMemcpyAsync pa\n");
	  cuerr = hipMemcpyAsync ( &(d_b[igpu][(ibuff%nstreams)*offset]), &pb[ibuff*offset], m*m*sizeof(double), hipMemcpyHostToDevice, myStreams[ibuff] );
	  cuerrmsg ( cuerr, "hipMemcpyAsync pb\n");
	  cuerr = hipMemcpyAsync ( &(d_c[igpu][(ibuff%nstreams)*offset]), &pc[ibuff*offset], m*m*sizeof(double), hipMemcpyHostToDevice, myStreams[ibuff] );
	  cuerrmsg ( cuerr, "hipMemcpyAsync pc\n");

	  // perform dgemm
	  custat = hipblasSetStream( cublasHandle[igpu], myStreams[ibuff] );
	  custatmsg ( custat, "hipblasSetStream \n");
	  custat = hipblasDgemm ( cublasHandle[igpu], HIPBLAS_OP_T, HIPBLAS_OP_T, m, m, m, &alpha, 
				 &(d_a[igpu][(ibuff%nstreams)*offset]), m, 
				 &(d_b[igpu][(ibuff%nstreams)*offset]), m, &beta, 
				 &(d_c[igpu][(ibuff%nstreams)*offset]), m ); 
	  custatmsg ( custat, "hipblasDgemm \n");
	  prowtile[ibuff] = irowtile;
	  pcoltile[ibuff] = icoltile;

	  // copy result back to host
	  cuerr = hipMemcpyAsync ( &pc[ibuff*offset], &(d_c[igpu][(ibuff%nstreams)*offset]), m*m*sizeof(double), hipMemcpyDeviceToHost, myStreams[ibuff] );
	  cuerrmsg ( cuerr, "hipMemcpyAsync pc\n");

	  // recored event to signal when D2H copy of result tile is complete
	  cuerr = hipEventRecord ( bufferfilled[ibuff], myStreams[ibuff] );
	  cuerrmsg ( cuerr, "hipEventRecord \n" );
	  
	  // update buffer / stream
	  ibuff++;
	  ibuff = ibuff%(nstreams*ngpus);
	  igpu = ibuff/nstreams;
	  itile++;

	}
      }
    }

    ibuff = 0;
    for ( itile=0; itile < nstreams*ngpus; itile ++ ) {

      cuerr = hipSetDevice ( ibuff/nstreams );
      cuerrmsg ( cuerr, "hipSetDevice pc\n");

      // make sure that buffers are filled
      hipEventSynchronize ( bufferfilled[ibuff] );
      cuerrmsg ( cuerr, "cudaEventSychronize \n" );

      // copy result in pinned buffer back to source 
      # pragma omp parallel for
      for ( int i=0; i<m; i++ ) {
	for ( int j=0; j<m; j++ ) {
	  c[(prowtile[ibuff]*m+i)*n+pcoltile[ibuff]*m+j] = pc[ibuff*offset+i*m+j];
	}
      }
	    
      ibuff++;
      ibuff = ibuff%(nstreams*ngpus);

    }

  }

  for ( int i = 0 ; i < ngpus; ++i) {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
    

  // record end time
  hipSetDevice(0);
  hipEventRecord (t_end,0);
  hipEventSynchronize(t_end);
  float et;
  hipEventElapsedTime (&et, t_start, t_end);
  
// check results
  printf ("\nchecking results: ");
  bool correct = true;
  double abs_error, sum_abs_errors = 0;
# pragma omp parallel for
  for ( int row = 0;  row < n; row++ ) {
    for ( int col = 0; col < n; col++ ) {
      
      abs_error = fabs(c[row * n + col] - a[row * n + col] );
      sum_abs_errors += abs_error;
      if (  abs_error > 10e-5 ) {
	printf ("FAILED\n\nerror: c[%d]: %f != a[%d]: %f", 
		row * n + col,  c[row * n + col], row * n + col,  a[row * n + col]);
	correct = false;
	break;
      }
    }
  }
  
  // report results
  if ( correct ) {
    printf ("SUCCESS");
    printf ("\nSum abs errors: %f", sum_abs_errors);
    printf("\nExecution time: %4.4f seconds\n", (double)et/1000.);     // cudaEventElapsedTime is in milliseconds
    printf(  "Gflop/s: %4.4f \n\n\n", 2.0e-6*n*n*n/et); // 2( * and + ) *n (inner dimension)*n^2(result size)/(time in ms.)
  } else {
    printf ("\nResult not correct, check your code !\n");
  }

  // clean up
  for ( int igpu=0; igpu<ngpus; igpu++ ) {
    hipSetDevice ( igpu );
    hipFree ( d_a[igpu] );
    hipFree ( d_b[igpu] );
    hipFree ( d_c[igpu] );
    hipblasDestroy ( cublasHandle[igpu] );
  }
  hipEventDestroy ( t_start  );
  hipEventDestroy ( t_end );
  hipHostFree ( pa );
  hipHostFree ( pb );
  hipHostFree ( pc );

  free (a);
  free (b);
  free (c);

}
