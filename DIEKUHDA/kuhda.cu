#include "kuhda.h"
/*
$$$$$$$\  $$$$$$\ $$$$$$$$\ $$\   $$\ $$\   $$\ $$\   $$\ $$$$$$$\   $$$$$$\      $$$$$$\   $$\   $$\ 
$$  __$$\ \_$$  _|$$  _____|$$ | $$  |$$ |  $$ |$$ |  $$ |$$  __$$\ $$  __$$\    $$  __$$\  $$ |  $$ |
$$ |  $$ |  $$ |  $$ |      $$ |$$  / $$ |  $$ |$$ |  $$ |$$ |  $$ |$$ /  $$ |   $$ /  \__| $$ |  $$ |
$$ |  $$ |  $$ |  $$$$$\    $$$$$  /  $$ |  $$ |$$$$$$$$ |$$ |  $$ |$$$$$$$$ |   $$ |		$$ |  $$ |
$$ |  $$ |  $$ |  $$  __|   $$  $$<   $$ |  $$ |$$  __$$ |$$ |  $$ |$$  __$$ |   $$ |		$$ |  $$ |
$$ |  $$ |  $$ |  $$ |      $$ |\$$\  $$ |  $$ |$$ |  $$ |$$ |  $$ |$$ |  $$ |   $$ |  $$\	$$ |  $$ |
$$$$$$$  |$$$$$$\ $$$$$$$$\ $$ | \$$\ \$$$$$$  |$$ |  $$ |$$$$$$$  |$$ |  $$ |$$\\$$$$$$  |	\$$$$$$  |
\_______/ \______|\________|\__|  \__| \______/ \__|  \__|\_______/ \__|  \__|\__|\______/	 \______/

Help: see https://docs.nvidia.com/cuda/cublas/index.html for specific help when using cuda */

// Other libraries and dependancies:
#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <omp.h>

/********************************************/
/* Allocation/deallocation on the HOST		*/
/********************************************/

/* kuhdaMallocV(unsigned long r): Allocates memory for a vector of length r
Arguments: r = length of vector
Return value: A pointer to a vector, or NULL if an error occured */
vector *kuhdaMallocV(unsigned long r){
	if (r <= 0){
		INPUT_ILL_ERR_LU(r);
		return NULL;
	}
	vector *out = (vector *) malloc(sizeof(*out));
	if (out == NULL) {
		MEM_ERR;
		free(out);
		return NULL;
	}
	out->r = r;
	out->data = (double *) calloc(r, sizeof(*out->data));
	if (out->data == NULL) {
		MEM_ERR;
		free(out->data);
		free(out);
		return NULL;
	}
	return out;
}


/* kuhdaFreeV(vector *freethisvector): free an allocated vector
Arguments: freethisvector = pointer to vector to be freed
Return value: NULL if an error occured */
void kuhdaFreeV(vector *freethisvector){
	printf("freeing this vector\n");
	if (freethisvector == NULL){
		INPUT_NULL_ERR;
	}
	free(freethisvector->data);
	free(freethisvector);
}


/* kuhdaMallocM(unsigned long r, unsigned long c):
Allocates memory for a matrix of length r*c. The matrix will be filled with zeros.
Remember that DIEKUHDA matrices (type matrix) are 1D arrays!
Arguments: r = number of matrix rows, c = number of matrix columns
Return value: A pointer to a matrix, or NULL if an error occured */
matrix *kuhdaMallocM(unsigned long r, unsigned long c){
	if (r <= 0 || c <=0 ){
		INPUT_ILL_ERR_LU(r);
		INPUT_ILL_ERR_LU(c);
		return NULL;
	}
	matrix *out = (matrix *) malloc(sizeof(*out));
	if (out == NULL) {
		MEM_ERR;
		free(out);
		return NULL;
	}
	out->r = r;
	out->c = c;
	out->data = (double *) calloc(r*c, sizeof(double));
	if (out->data == NULL) {
		MEM_ERR;
		free(out->data);
		free(out);
		return NULL;
	}
	return out;
}

/* kuhdaMallocM1(unsigned long r, unsigned long c):
Allocates memory for a matrix of length r*c. The matrix will be filled with ones.
Remember that DIEKUHDA matrices (type ccMatrix) are 1D arrays!
Arguments: r = number of matrix rows, c = number of matrix columns
Return value: A pointer to a matrix, or NULL if an error occured */
matrix *kuhdaMallocM1(unsigned long r, unsigned long c){
	matrix *out = kuhdaMallocM(r, c);
	unsigned long i, j;
	for (i = 0; i < r; ++i){
		for (j = 0; j < c; ++j){
			*(out->data + i*c + j) = 1.0;
		}
	}
	return out;
}

/* kuhdaMallocMdiag(unsigned long r, unsigned long c):
Allocates memory for a matrix of length r*c. The matrix will be a diagonal matrix.
Remember that DIEKUHDA matrices (type ccMatrix) are 1D arrays!
Arguments: r = number of matrix rows, c = number of matrix columns
Return value: A pointer to a matrix, or NULL if an error occured */
matrix *kuhdaMallocMdiag(unsigned long r, unsigned long c){
	matrix *out = kuhdaMallocM(r, c);
	unsigned long i;
	for (i = 0; i < r*c; i += c + 1){
		*(out->data + i) = 1.0;
	}
	return out;
}


/* kuhdaFreeM(matrix *freethismatrix): free an allocated matrix
Arguments: freethismatrix = pointer to matrix to be freed, type = cuda or kuhda
Return value: NULL if an error occured */
void kuhdaFreeM(matrix *freethismatrix, char type){
	if (freethismatrix == NULL) INPUT_NULL_ERR;

	switch(type) {

   	case 'c': // a kuhda matrix with data member on a device
      	GPUCHECK(hipFree(freethismatrix->data));
		free(freethismatrix);
      	break;

	case 'p': // a kuhda matrix with data member pinned on the host
    	GPUCHECK(hipHostFree(freethismatrix->data));
		GPUCHECK(hipHostFree(freethismatrix));
      	break;

   	case 'k': // a kuhda matrix with data member on the host
   		free(freethismatrix->data);
   		free(freethismatrix);
		break;
	}
}


/********************************************/
/*    Vector / Matrix printing utilities    */
/********************************************/
void kuhdaPrintV(vector *printthisvector){
	if (printthisvector == NULL){
		INPUT_NULL_ERR;
	}
	unsigned long i;
	printf("[");
	for (i = 0; i < printthisvector->r; ++i){
		printf("%5.3lf", printthisvector[i]);
	}
	printf("]\n");
}


void kuhdaPrintM(matrix *printthismatrix){
	if (printthismatrix == NULL){
		INPUT_NULL_ERR;
	}
	unsigned long i,j;
	for (i = 0; i < printthismatrix->r; ++i){
		printf("|");
		for (j = 0; j < printthismatrix->c; ++j){
			printf("%6.2lf", printthismatrix->data[i*printthismatrix->c + j]);
		}
		printf("|\n");
	}
	printf("\n");
}

void kuhdaPrintDeviceM(matrix *printthismatrix){
	size_t matrixsize = (size_t) printthismatrix->r * printthismatrix->c * sizeof(double);
	double *printme = (double*)malloc(matrixsize);
		hipMemcpy(printme, printthismatrix->data, matrixsize, hipMemcpyDeviceToHost);
		unsigned long i,j;
	for (i = 0; i < printthismatrix->r; ++i){
		printf("|");
		for (j = 0; j < printthismatrix->c; ++j){
			printf("%6.2lf", printme[i*printthismatrix->c + j]);
		}
		printf("|\n");
	}
	printf("\n");
}

// Test whether all elements of this matrix are equal to its' dimensions.
// Only for the result of multiplication on square ones!
void kuhdaTestM(unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, matrix *testhismatrix){
	if (testhismatrix == NULL){
		INPUT_NULL_ERR;
	}
	if (rowstart - rowstop != colstart - colstop){
		printf("The testfunction is only deigned for square tiles.");
		return;
	}

	unsigned long i,j,value = 0,as_we_would_expect = (int)(rowstop - rowstart);
	for (i=rowstart; i<rowstop; ++i){
		for (j=colstart; j<colstop; ++j){
			value = (int)testhismatrix->data[i*testhismatrix->c + j];
			if (value != as_we_would_expect){
				printf("The matrix does not contain the expected results at ");
				printf("(%d, %d) = %d != %d\n", i,j, value, as_we_would_expect);
				FAIL_ERR(value);
				return;
			}
		}
	}
	printf("Test succeeded. No errors.\n");
}

// Test whether all elements of this matrix are equal to its' dimensions.
// Only for the result of multiplication on square ones!
int kuhdaTestMsilent(unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, matrix *testhismatrix, int verbose){
	if (testhismatrix == NULL){
		INPUT_NULL_ERR;
		return -1;
	}
	if (rowstart - rowstop != colstart - colstop){
		if (verbose != 0) printf("The testfunction is only deigned for square tiles.");
		return -1;
	}

	unsigned long i, j, value = 0, as_we_would_expect = (int)(rowstop - rowstart);
	for (i = rowstart; i < rowstop; ++i){
		for (j = colstart; j < colstop; ++j){
			value = (int)testhismatrix->data[i*testhismatrix->c + j];
			if (value != as_we_would_expect){
				if (verbose != 0) printf("The matrix does not contain the expected results at (%d, %d) = %d != %d\n", i,j, value, as_we_would_expect);
				FAIL_ERR(value);
				return -1;
			}
		}
	}
	if (verbose != 0) printf("Test succeeded. No errors.\n");
	return 0;
}



/********************************************/
/* Allocation/deallocation on the DEVICE(S) */
/********************************************/

/* kuhdaMallocDeviceM: hipMalloc of a [r * c] matrix structure on the device */
matrix *kuhdaMallocDeviceM(unsigned long r, unsigned long c){
	if (r <= 0){
        INPUT_ILL_ERR_LU(r);
        return NULL;
    }
    if (c <= 0){
        INPUT_ILL_ERR_LU(c);
        return NULL;
    }

    matrix *out = (matrix *) malloc(sizeof(*out));
    if (out == NULL) {
			MEM_ERR;
			GPUCHECK(hipFree(out));
			return NULL;
		}

	out->r = r;
	out->c = c;
    out->data = NULL;
	GPUCHECK(hipMalloc((void**)&out->data, r*c*sizeof(double)));
    if (out->data == NULL) {
		MEM_ERR;
		GPUCHECK(hipFree(out->data));
	    GPUCHECK(hipFree(out));
		return NULL;
	}
	return out;
}


/* PINNED allocation routine for matrix of dimension [r * c] */
matrix *kuhdaMallocMP(unsigned long r, unsigned long c){
	if (r <= 0){
        INPUT_ILL_ERR_LU(r);
        return NULL;
    }
    if (c <= 0){
        INPUT_ILL_ERR_LU(c);
        return NULL;
    }

    matrix *out = NULL;
    GPUCHECK(hipHostMalloc((void**)&out, sizeof(*out)));
    if (out == NULL) {
		MEM_ERR;
		GPUCHECK(hipHostFree(out));
		return NULL;
	}

	out->r = r;
	out->c = c;
    out->data = NULL;
	GPUCHECK(hipHostAlloc((void**)&out->data, r*c*sizeof(double), hipHostMallocPortable));
    if (out->data == NULL) {
		MEM_ERR;
		GPUCHECK(hipHostFree(out->data));
	    GPUCHECK(hipHostFree(out));
		return NULL;
	}
	return out;
}

/* PINNED allocation for [r * c] matrix of ones */
matrix *kuhdaMallocMP1(unsigned long r, unsigned long c){
	matrix *out = kuhdaMallocMP(r, c);
	unsigned long i, j;
	for (i = 0; i < r; ++i){
		for (j = 0; j < c; ++j){
			*(out->data + i*c + j) = 1.0;
		}
	}
	return out;
}

/* PINNED allocation for [r * c] identity matrix */
matrix *kuhdaMallocMdiagP(unsigned long r, unsigned long c){
	matrix *out = kuhdaMallocMP(r, c);
	unsigned long i;
	for (i = 0; i < r*c; i += c + 1){
		*(out->data + i) = 1.0;
	}
	return out;
}


// Fill and test with value
void kuhdaFillWithValue(matrix *A, double value){
	unsigned long i, j;
	for (i = 0; i < A->r; ++i){
		for (j = 0; j < A->c; ++j){
			A->data[i*A->c + j] = value;
		}
	}
}

void kuhdaTestForValue(matrix *A, double value, int verbose){
	unsigned long i, j;
	int result = 0;
	for (i = 0; i < A->r; ++i){
		for (j = 0; j < A->c; ++j){
			if (A->data[i*A->c + j] != value){
				fprintf(stderr,"%s: encountered wrong value %.2lf instead of %.2lf at (%zu,%zu)\n",__func__, A->data[i*A->c + j], value, i, j);
				result = -1;
				return;
			}
		}
	}
	if (result == 0 && verbose == 1) printf("%s tested correctly for value %.2lf\n", __func__, value);
}

void kuhdaFillDiagonalWithValue(matrix *A, double value){
	unsigned long i;
	for (i = 0; i < A->r*A->c; i += A->c + 1) A->data[i] = value;
}

void kuhdaTestDiagonalForValue(matrix *A, double value, int verbose){
	unsigned long i;
	int result = 0;
	for (i = 0; i < A->r*A->c; i += A->c + 1){
		if (A->data[i] != value){
			fprintf(stderr,"%s: encountered wrong value %.2lf instead of %.2lf at (%zu,%zu)\n",__func__, A->data[i], value, i, i);
			result = -1;
			return;
		}
	}
	if (result == 0 && verbose == 1) printf("%s tested correctly for value %.2lf\n", __func__, value);
}

/********************************************/
/* 				 Data transfers 			*/
/********************************************/

/*
TileHostToGPU: memcopy tile of host matrix to device asynchronously.
Arguments: dimensions / location of tile to be copied, pointers to hostmatrix & device-tile, streams
Return value: none
*/
void TileHostToGPU(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
					matrix *h_matrix, matrix *d_tile, hipStream_t stream )
{	
	// check input
	if (h_matrix == NULL || d_tile == NULL) 	INPUT_NULL_ERR;
	if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
	if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
	if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
	if (stream == NULL) INPUT_NULL_ERR;

	unsigned long cols = colstop - colstart, i, j;
	hipError_t failure;

	// allocate space (size of a single tile row) on the host:
	// double *memacc = (double*)malloc(cols*sizeof(double));
	double *memacc = NULL;
	GPUCHECK(hipHostMalloc((void**)&memacc, cols*sizeof(double)));
	if (memacc == NULL){
		MEM_ERR;
		//free(memacc);
		hipHostFree(memacc);
		return;
	}

	// 'strided' copy, row by row
	for (i=rowstart; i<rowstop; ++i){
		for (j=colstart; j<colstop; ++j){
			// fill memacc with host-matrix data one (tile-)row at a time:
			memacc[j-colstart] = h_matrix->data[i * h_matrix->c + j];
		}
		GPUCHECK(hipStreamSynchronize(stream));
		// Asynchronous copy to device
		// takes (d_arr, h_arr, nbytes, hipMemcpyHostToDevice, stream)
		// failure = GPUCHECK(hipMemcpyAsync((void*) (&d_tile->data[0] + (cols * (i-rowstart))), memacc, cols*sizeof(double), hipMemcpyHostToDevice, stream));
		failure = GPUCHECK(hipMemcpy((void*) (&d_tile->data[0] + (cols * (i-rowstart))), memacc, cols*sizeof(double), hipMemcpyHostToDevice));
		
		GPUCHECK(hipStreamSynchronize(stream));
		
		if (failure != 0) {
			FAIL_ERR(failure);
			hipFree(d_tile);
			}
	}
	hipHostFree(memacc);
	// free(memacc);
	return;
}


/*
TileGPUToHost: memcopy tile of device matrix to host asynchronously.
Arguments: dimensions / location of tile to be copied, pointers to hostmatrix & device-tile, streams
Return value: none
*/
void TileGPUToHost(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
					matrix *d_tile, matrix *h_matrix, hipStream_t stream )
{
	// check input
	if (h_matrix == NULL || d_tile == NULL) 	INPUT_NULL_ERR;
	if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
	if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
	if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
	if (stream == NULL) INPUT_NULL_ERR;


	unsigned long cols = colstop - colstart, i, j;
	hipError_t failure;

	//double *memacc = (double*)malloc(cols*sizeof(double));
	double *memacc = NULL;
	GPUCHECK(hipHostMalloc((void**) &memacc, cols*sizeof(double)));
	if (memacc == NULL){
		MEM_ERR;
		//free(memacc);
		hipHostFree(memacc);
		return;
	}

	// 'strided' copy, row by row
	for (i=rowstart; i<rowstop; ++i){
		// takes (d_arr, h_arr, nbytes, hipMemcpyHostToDevice, stream)
		failure = GPUCHECK(hipMemcpyAsync(memacc, (void*) (&d_tile->data[0] + (cols * (i-rowstart))), cols*sizeof(double), hipMemcpyDeviceToHost, stream));
		for (j=colstart; j<colstop; ++j){
			h_matrix->data[i * h_matrix->c + j] = memacc[j-colstart];
			//memacc[j-colstart] = h_matrix->data[i * h_matrix->c + j];
		}

		if (failure != 0) {
			FAIL_ERR(failure);
			hipFree(d_tile);
		}
	}
	hipHostFree(memacc);
	return;
}

/*
TileGPUAddToHost: memcopy and add tile of device matrix to host.
Arguments: dimensions / location of tile to be copied, pointers to hostmatrix & device-tile, streams
Return value: none
*/
void TileGPUAddToHost(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
					  	matrix *d_tile, matrix *h_matrix, hipStream_t stream )
{
	// check input
	if (h_matrix == NULL || d_tile == NULL) 	INPUT_NULL_ERR;
	if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
	if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
	if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
	if (stream == NULL) INPUT_NULL_ERR;


	unsigned long cols = colstop - colstart, i, j;
	hipError_t failure;

	// double *memacc = (double*)malloc(cols*sizeof(double));
	double *memacc = NULL;
	GPUCHECK(hipHostMalloc(&memacc, cols*sizeof(double)));

	if (memacc == NULL){
		MEM_ERR;
		//free(memacc);
		hipHostFree(memacc);
		return;
	}

	// 'strided' copy, row by row
	for (i=rowstart; i<rowstop; ++i){
		GPUCHECK(hipStreamSynchronize(stream));
		// takes (d_arr, h_arr, nbytes, hipMemcpyHostToDevice, stream)
		// failure = GPUCHECK(hipMemcpyAsync(memacc, (void*) (&d_tile->data[0] + (cols * (i-rowstart))), cols*sizeof(double), hipMemcpyDeviceToHost, stream));
		failure = GPUCHECK(hipMemcpy((void*) (&d_tile->data[0] + (cols * (i-rowstart))), memacc, cols*sizeof(double), hipMemcpyHostToDevice));
		for (j=colstart; j<colstop; ++j){
			h_matrix->data[i * h_matrix->c + j] += memacc[j-colstart];
			//memacc[j-colstart] = h_matrix->data[i * h_matrix->c + j];
		}
		GPUCHECK(hipStreamSynchronize(stream));
		if (failure != 0) {
			FAIL_ERR(failure);
			hipFree(d_tile);
		}
	}
	hipHostFree(memacc);
	// free(memacc);
	return;
}



/****************************************/
/* 				Utilities				*/
/****************************************/

/*gpuAssert(hipError_t code, const char *file, int line): check for cuda errors.
Arguments: code = cudafunction to be wrapped around, file and line = place where the error occured */
hipError_t gpuAssert(hipError_t code, const char *file, int line){
	if (code != hipSuccess){
		fprintf(stderr, "GPUassert: error in file %s, line %d\n", file, line);
  		fprintf(stderr,"code %d with reason %s\n", code, hipGetErrorString(code));
    	exit(1);
   }
   return code;
}

// New definitions for error checks:
hipblasStatus_t cublasAssert(hipblasStatus_t error, const char *file, int line){
    if (error != HIPBLAS_STATUS_SUCCESS){
      fprintf(stderr, "CUBLASCHECK: error in file %s, line %d \n", file, line);
      fprintf(stderr,"error code = %d\n", error);
      exit(1);
	}
	return error;
}


// GPU warmup function for timing experiments
void kuhdaWarmup(int devicecount){
	// Sync current device
	hipDeviceSynchronize();
	int device;
	// #pragma omp parallel for private(device) num_threads(devicecount)
	for(device = 0; device < devicecount; ++device){
		GPUCHECK(hipSetDevice(device));
		int *testint = 0;
		GPUCHECK(hipMalloc((void**)&testint,sizeof(int)));
		GPUCHECK(hipFree(testint));
		GPUCHECK(hipDeviceSynchronize());
	}
}

void kuhdaWarmupDevice(int device){
	// Sync current device
	GPUCHECK(hipSetDevice(device));
	// Allocate space for a dummy int
	int *testint = 0;
	GPUCHECK(hipMalloc((void**)&testint,sizeof(int)));
	GPUCHECK(hipFree(testint));
	GPUCHECK(hipDeviceSynchronize());
}

// Check available memory to reduce tile size when too large
size_t kuhdaAvailableMemoryOnCurrentDevice(){
	size_t memfree, memtotal;
	GPUCHECK(hipMemGetInfo(&memfree, &memtotal));
	return memfree;
}


/********************************************/
/* 			Necessary computations			*/
/********************************************/

/* kuhdaTimeDGEMM(unsigned long m, unsigned long n, unsigned long k): compute the number of
floating point operations per second, as performed by hipblasDgemm.
C <- alpha * AB + beta*C	 with	 [A] = m x k, [B] = k x n, [C] = m x n

Arguments: m, n, k = formal dimensions of the matrices A, B and C,
time_diff = the time it took to perform the computations with hipblasDgemm,
verbose = whether we want to print the output on the console ('0' = nothing prints, '1' = results will be printed)

Return value: the number of GigaFlops (GFLOPS), or NULL if an error occured */
long long kuhdaTimeDGEMM(matrix *d_matrix, int reps, int verbose){
	if (d_matrix == NULL){
		INPUT_NULL_ERR;
		return -1;
	}
	// Data for the computations:
	unsigned int m = d_matrix->r, k = d_matrix->r, n = d_matrix->c;
	double alpha = 1.0, beta  = 0.0;
	hipblasHandle_t handle;
	int failure = hipblasCreate(&handle);
	if (failure != 0){
		FAIL_ERR(failure);
		return -1;
	}
	hipStream_t stream = (hipStream_t) malloc(sizeof(hipStream_t));
  	GPUCHECK(hipStreamCreate(&stream));
	failure = hipblasSetStream(handle, stream);
	if (failure != 0){
		FAIL_ERR(failure);
		return -1;
	}

	// Events for the dgemm timing:
	hipEvent_t start, stop;
	GPUCHECK(hipEventCreate(&start));
	GPUCHECK(hipEventCreate(&stop));

	int rep = 0;
	GPUCHECK(hipEventRecord(start, 0));
	GPUCHECK(hipStreamSynchronize(0));
	for (rep = 0; rep < reps; ++rep){
		failure = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
			d_matrix->data, m, d_matrix->data, k, &beta, d_matrix->data, m);
		if (failure != 0){
			FAIL_ERR(failure);
			return -1;
		}
	}
	GPUCHECK(hipStreamSynchronize(0));
  //GPUCHECK(hipDeviceSynchronize()); // Not necessary when using cudaEvents
  GPUCHECK(hipEventRecord(stop, 0));
	GPUCHECK(hipEventSynchronize(stop));

	float milliseconds = 0;
	GPUCHECK(hipEventElapsedTime(&milliseconds, start, stop));

	// Number of computations was found here:
	// https://devtalk.nvidia.com/default/topic/482834/how-to-compute-gflops-for-gemm-blas/
	long int numerator    = (long int)(m * n) * (2 * ((long long)k) + 2) * reps;
	long long denominator = 1.0e6 * milliseconds;
	long long gflops = numerator / denominator;
	if (verbose !=0){
		printf("%lu GFLPS\n", gflops);
	}
	// Clean up:
	hipblasDestroy(handle);
	GPUCHECK(hipEventDestroy(start));
	GPUCHECK(hipEventDestroy(stop));
	return gflops;
}



/* kuhdamm(matrix *d_A_tile, matrix *d_B_tile, matrix *d_C_tile, int verbose):
perform matrix-matrix multiplication of tiles on a device, performed by hipblasDgemm.
C <- alpha * AB + beta*C	 with	 [A] = m x k, [B] = k x n, [C] = m x n

Arguments: m, n, k = formal dimensions of the matrices A, B and C,
verbose = whether we want to print the output on the console
('0' = nothing prints, '1' = results will be printed)

Return value: the number of GigaFlops (GFLOPS), or NULL if an error occured */
int kuhdamm(matrix *d_A_tile, matrix *d_B_tile, matrix *d_C_tile, hipStream_t stream, hipblasHandle_t handle){
	if (d_A_tile == NULL || d_B_tile == NULL || d_C_tile == NULL){
		INPUT_NULL_ERR;
		return -1;
	}
	if (d_A_tile->r != d_C_tile->r || d_A_tile->c != d_B_tile->r || d_B_tile->c != d_C_tile->c){
		INPUT_ILL_ERR_D(d_A_tile->r);
		return DIEKUHDA_DIMENSION_MISMATCH;
	}
	if (stream == NULL) INPUT_NULL_ERR;

	// Data for the computations:
	unsigned int m = d_A_tile->r, k = d_A_tile->c, n = d_C_tile->c;
	double alpha = 1.0, beta  = 0.0;
	// hipblasHandle_t handle;
	// int failure = hipblasCreate(&handle);
	// if (failure != 0){
	// 	FAIL_ERR(failure);
	// 	return -1;
	// }

	CUBLASCHECK(hipblasSetStream(handle, stream));
	// if (failure != 0){
	// 	FAIL_ERR(failure);
	// 	return -1;
	// }
	CUBLASCHECK(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
			d_A_tile->data, m, d_B_tile->data, k, &beta, d_C_tile->data, m));
	// if (failure != 0){
	// 	FAIL_ERR(failure);
	// 	return -1;
	// }
	// hipblasDestroy(handle);
	return 0;
}




//////////////////////////////////////////////////////////////////
// DEPRECATED - use allocation and data transfers separately... //
//////////////////////////////////////////////////////////////////

/* kuhdaMatrixToGPU(matrix *h_matrix): allocate a matrix on the device and copy contents of host matrix.
Arguments: rows, cols = which tile of rows x cols is taken from the host matrix
Return value: NULL if an error occured */
matrix *kuhdaMatrixToGPU(unsigned long rows, unsigned long cols, matrix *h_matrix){
	if (h_matrix == NULL){
		INPUT_NULL_ERR;
	}

	hipError_t failure;
	matrix *d_matrix = kuhdaMallocM(rows, cols);
	// failure = GPUCHECK(hipMalloc(&d_matrix->data, rows*cols*sizeof(double)));
	failure = GPUCHECK(hipMalloc((void**)&d_matrix->data, rows*cols*sizeof(double))); // Tip from HH
	if (failure != 0) {
		MEM_ERR;
		kuhdaFreeM(d_matrix, 'k');
	} // rows, cols = which tile of rows x cols is taken from the host matrix

	//failure = hipblasSetMatrix(rows, cols, sizeof(double*), h_matrix->data, rows, d_matrix->data, rows);
	//failure = hipMemcpy2D(&h_matrix->data, &d_matrix->data, rows*cols*sizeof(double), hipMemcpyHostToDevice);
	failure = GPUCHECK(hipMemcpy(d_matrix->data, h_matrix->data, rows*cols*sizeof(double), hipMemcpyHostToDevice));
	if (failure != 0) {
		FAIL_ERR(failure);
		hipFree(d_matrix);
	}

	return d_matrix;
}

/* kuhdaTileToGPU(matrix *h_matrix): allocate a matrix on the device and copy contents of host matrix.
Arguments: rows, cols = which tile of rows x cols is taken from the host matrix
Return value: NULL if an error occured */
double *kuhdaTileToGPU(unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, matrix *h_matrix){
	if (h_matrix == NULL) 	INPUT_NULL_ERR;
	if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
	if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);

	unsigned long rows = rowstop - rowstart, cols = colstop - colstart;
	unsigned long i, j;

	double *memacc = (double*)malloc(cols*sizeof(double));
	if (memacc == NULL){
		MEM_ERR;
	}
	hipError_t failure;
	// matrix *d_tile = kuhdaMallocM(rows, cols);
	double *d_tile = NULL;

	// failure = GPUCHECK(hipMalloc(&d_matrix->data, rows*cols*sizeof(double)));
	failure = GPUCHECK(hipMalloc((void**)&d_tile, rows*cols*sizeof(double))); // Tip from HH
	if (failure != 0) {
		MEM_ERR;
		hipFree(d_tile);
	//	kuhdaFreeM(d_matrix, 'k');
	} // rows, cols = which tile of rows x cols is taken from the host matrix

	//double* tilep = &d_tile[0];
	// 'strided' copy
	for (i=rowstart; i<rowstop; ++i){
		for (j=colstart; j<colstop; ++j){
				memacc[j-colstart] = h_matrix->data[i * h_matrix->c + j];
		}
		//printf("%zu\n", cols * (i-rowstart) );
		// printf("%zu\n", sizeof( *(tilep + (cols * (i-rowstart))) ) );
		//failure = GPUCHECK(hipMemcpy((void*)d_tile + (cols * (i-rowstart)), memacc, cols*sizeof(double), hipMemcpyHostToDevice));
		failure = GPUCHECK(hipMemcpy((void*) (&d_tile[0] + (cols * (i-rowstart))), memacc, cols*sizeof(double), hipMemcpyHostToDevice));

		if (failure != 0) {
			FAIL_ERR(failure);
			hipFree(d_tile);
		}
	}

	return d_tile;
}


void kuhdaMatrixToHost(unsigned long rows, unsigned long cols, matrix *d_matrix, matrix *h_matrix){
	if (h_matrix == NULL || d_matrix == NULL){
			INPUT_NULL_ERR;
	}
	//int failure = hipblasGetMatrix(rows, cols, sizeof(double), d_matrix->data, d_matrix->r, h_matrix->data, h_matrix->r);
	//int failure = hipMemcpy2D(h_matrix->data, d_matrix->data, rows*cols*sizeof(double), hipMemcpyDeviceToHost);
	hipError_t failure = GPUCHECK(hipMemcpy(h_matrix->data, d_matrix->data, rows*cols*sizeof(double), hipMemcpyDeviceToHost));
	if (failure != 0){
		FAIL_ERR(failure);
	}
}


void kuhdaTileToHost(unsigned long rows, unsigned long cols, double *d_tile, matrix *h_matrix){
	if (h_matrix == NULL || d_tile == NULL) INPUT_NULL_ERR;
	if (rows != h_matrix->r) INPUT_ILL_ERR_LU(rows);
	if (cols != h_matrix->c) INPUT_ILL_ERR_LU(cols);

	//int failure = hipblasGetMatrix(rows, cols, sizeof(double), d_matrix->data, d_matrix->r, h_matrix->data, h_matrix->r);
	//int failure = hipMemcpy2D(h_matrix->data, d_matrix->data, rows*cols*sizeof(double), hipMemcpyDeviceToHost);
	hipError_t failure = GPUCHECK(hipMemcpy(h_matrix->data, d_tile, rows*cols*sizeof(double), hipMemcpyDeviceToHost));
	if (failure != 0){
		FAIL_ERR(failure);
	}
}

/********************************************/
/* 				cuda-specific		  		*/
/********************************************/
/* kuhdaMilkCan(int streamnums):
Arguments: number of streams
Return value: euter with strams and handles, or NULL if an error occured */
/*
can *kuhdaMilkCan(int streamnums){
	if (streamnums <= 0){
		INPUT_ILL_ERR_D(streamnums);
		return NULL;
	}
	can *mm = (can *) malloc(sizeof(*mm));
	if (mm == NULL) {
		MEM_ERR;
		free(mm);
		return NULL;
	}
	int failure;
	failure = hipblasCreate(&(mm->handle));
	if (failure != 0){
		FAIL_ERR(failure);
		return NULL;
	}
	mm->streams = (hipStream_t *) malloc(streamnums*sizeof(hipStream_t));
	if (mm->streams == NULL) {
		MEM_ERR;
		free(mm->streams);
		free(mm);
		return NULL;
	}
	int i;
	for (i = 0; i < streamnums; ++i){
		failure = hipStreamCreate(&(mm->streams)[i]);
		if (failure != 0){
			FAIL_ERR(failure);
			return NULL;
		}
	}
	return mm;
}
*/
