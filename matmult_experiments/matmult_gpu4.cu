#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hipblas.h>
#include "matmult_transfer_gpu.h"
#define CHECK checkCudaErrors

extern "C" {

    // SINGLE_GPU split A and B version - hiding overlap.
	void matmult_gpu4(int m, int n, int k, double *A_in, double *B, double *C_in)
	{
        omp_set_nested(1);

        int numSplits = 4;
        int numDevices = 1;
        //hipGetDeviceCount(&numDevices);
        int numSplitsPerDevice = numSplits / numDevices;

        int lda = k;
        int ldb = n;
        int ldc = n;
		long size_A = m * lda * sizeof(double);
        long size_B = k * ldb * sizeof(double);
        long size_C = m * ldc * sizeof(double);

        m /= numDevices;
        size_A /= numDevices;
        size_C /= numDevices;

        //#pragma omp parallel for firstprivate(m, size_A, size_C)
        for (int device = 0; device < numDevices; device++) {

            hipSetDevice(device);

            double *A = A_in + m * lda * device;
            double *C = C_in + m * ldc * device;

            hipStream_t stream[numSplitsPerDevice];
            hipEvent_t event[numSplitsPerDevice];
            for (int split_m = 0; split_m < numSplitsPerDevice; split_m++) {
                hipStreamCreate(&stream[split_m]);
                hipEventCreate(&event[split_m]);
            }

            // Allocate on device
            double *d_A, *d_B, *d_C;
            allocate_on_gpu(m, n, k, &d_A, &d_B, &d_C);
            //CHECK(hipMemset(d_C, 2, size_C));

            m /= numSplitsPerDevice;
            size_A /= numSplitsPerDevice;
            size_C /= numSplitsPerDevice;
            n /= numSplits;
            size_B /= numSplits;

            hipblasHandle_t handle;
            hipblasCreate(&handle);

            //#pragma omp parallel for
            for (int split_m = 0; split_m < numSplits; split_m++) {

                //hipSetDevice(device);
                CHECK(hipMemcpyAsync(d_A + m * lda * split_m, A + m * lda * split_m, size_A, hipMemcpyHostToDevice, stream[split_m]));
                CHECK(hipMemcpy2DAsync(d_B + n * split_m, ldb * sizeof(double), B + n * split_m, ldb * sizeof(double), n * sizeof(double), k, hipMemcpyHostToDevice, stream[split_m]));
                CHECK(hipEventRecord(event[split_m], stream[split_m]));
            }

            for (int split_m = 0; split_m < numSplits; split_m++) {
                for (int split_n = 0; split_n < numSplits; split_n++) {

                    CHECK(hipStreamWaitEvent(stream[split_m], event[split_n], 0));

                    double time0 = omp_get_wtime();
                    const double alpha = 1.0;
                    const double beta = 0.0;
                    hipblasSetStream(handle, stream[split_m]);
                    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B + n * split_n, ldb, d_A + m * lda * split_m, lda, &beta, d_C + m * ldc * split_m + n * split_n, ldc);
                    CHECK(hipMemcpy2DAsync(C + m * ldc * split_m + n * split_n, ldc * sizeof(double), d_C + m * ldc * split_m + n * split_n, ldc * sizeof(double), n * sizeof(double), m, hipMemcpyDeviceToHost, stream[split_m]));
                    //printf("Computing C = A * B           | %5.4f s\n", omp_get_wtime() - time0);
                }
            }
            for (int split_m = 0; split_m < numSplits; split_m++) {
                CHECK(hipStreamSynchronize(stream[split_m]));
            }
            hipblasDestroy(handle);
            free_on_gpu(d_A, d_B, d_C);
        }
    }
}
