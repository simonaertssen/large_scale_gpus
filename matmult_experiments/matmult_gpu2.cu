#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hipblas.h>
#include "matmult_transfer_gpu.h"

extern "C" {

    // SINGLE_GPU split version - hiding overlap.
	void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C)
	{
        hipSetDevice(0);

		// Allocate on device
        double *d_A, *d_B, *d_C;
        allocate_on_gpu(m, n, k, &d_A, &d_B, &d_C);
		long size_B = k * n * sizeof(double);
		long size_C = m * n * sizeof(double);

		// Transfer B to device
		checkCudaErrors(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemset(d_C, 0, size_C));

        int numSplits = 8;
        m /= numSplits;
        long elms = m * n;
		long size_A = m * k * sizeof(double);
        size_C /= numSplits;
 
#pragma omp parallel for
        for (int split = 0; split < numSplits; split++) {

            hipSetDevice(0);
            hipblasHandle_t handle;
            hipblasCreate(&handle);
            checkCudaErrors(hipMemcpyAsync(d_A + m * k * split, A + m * k * split, size_A, hipMemcpyHostToDevice));
            double time0 = omp_get_wtime();

            const double alpha = 1.0;
            const double beta = 0.0;
            hipblasSetStream(handle, hipStreamPerThread);
            hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A + m * k * split, k, &beta, d_C + elms * split, n); // Row major.
            checkCudaErrors(hipMemcpyAsync(C + elms * split, d_C + elms * split, size_C, hipMemcpyDeviceToHost, hipStreamPerThread));
            hipStreamSynchronize(hipStreamPerThread);
            time0 = omp_get_wtime() - time0; printf("Computing C = A * B           | %5.4f s %5.4f Gflops\n", time0, 2.0 * m * n * k * 1e-9 / time0);

            hipblasDestroy(handle);
        }

        free_on_gpu(d_A, d_B, d_C);
	}
}
