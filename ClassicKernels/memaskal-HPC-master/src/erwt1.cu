#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipblas.h>
#include "gpu_timer.h"

#define DEBUG 0
#define THREADS 32
#define DEVIDE_ID 0
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// Debug mode only
int PRINT = 0;

// Global vars
GpuTimer timer;
double *d_A, *d_C, *h_A, *h_C;

// Utility functions declaration
void cleanUp();
void gpuAssert(hipError_t, const char *, int);
void print_matrix(const double * const, const int, const int);

// Kernel wrapper
void gpu_mul(const double *, double *, const int, const int);

// Kernel declarations
__global__ void fill_matrix(double * const, const int, const int);
__global__ void matrix_equals_to(const double *, const double, const int, const int);

int main(int argc, char *argv[]) {
		
	int rows_A, cols_A, rows_C, cols_C;

	if (argc < 2) {
		fprintf(stderr, "Please provide rows, cols of matrix A\n");
		exit(1);
	}
	 
	rows_A = atoi(argv[1]);
	cols_A = atoi(argv[2]);
	rows_C = cols_C = cols_A;		// A**T*A = C
	
	// debug only
	if (argc > 3) {
		PRINT = atoi(argv[3]);
	}
	
	// Set cuda device
	gpuErrchk(hipSetDevice(DEVIDE_ID));
	
	h_A = h_C = NULL;
	d_A = d_C = NULL;
	
	if (PRINT) {
		// Allocate A, C matrices on Host
		if ((h_A = (double *)malloc(rows_A * cols_A * sizeof(double))) == NULL ||
			(h_C = (double *)malloc(rows_C * cols_C * sizeof(double))) == NULL) {
			fprintf(stderr, "Host allocation error\n");
			cleanUp();
			exit(1);
		}
	}

	// Allocate A, C matrices on Device
	gpuErrchk(hipMalloc(&d_A, rows_A * cols_A * sizeof(double)));
	gpuErrchk(hipMalloc(&d_C, rows_C * cols_C * sizeof(double)));
	
	dim3 block(THREADS, THREADS);
	dim3 grid(ceil(((float)cols_A)/block.x), ceil(((float)rows_A)/block.y));
	fill_matrix<<<grid, block>>>(d_A, rows_A, cols_A);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	if (PRINT) {
		printf("A =\n");
		hipMemcpy(h_A, d_A, rows_A * cols_A * sizeof(double), hipMemcpyDeviceToHost);
		print_matrix(h_A, rows_A, cols_A);
	}

	// Perform the multiplications	
	gpu_mul(d_A, d_C, rows_A, cols_A);
	
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
			
	if (PRINT) {
		printf("C =\n");
		hipMemcpy(h_C, d_C, rows_C * cols_C * sizeof(double), hipMemcpyDeviceToHost);
		print_matrix(h_C, rows_C, cols_C);
	}
	printf("Time elapsed: %f ms\n", timer.Elapsed());

	#if DEBUG
	double value = 0.;
	for (int i = 1; i < rows_A; ++i) {
		value += i * i;
	}	
	matrix_equals_to<<<grid, block>>>(d_C, value, rows_C, cols_C);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	#endif
	
	cleanUp();
	return 0;
}


/**
* Fill matrix sets the value of each element in matrix A to it's row number
* for debugging purposes
**/
__global__ void fill_matrix(double * const A, const int rows, const int cols) {
	
	const int col = blockIdx.x * blockDim.x + threadIdx.x,
			  row = blockIdx.y * blockDim.y + threadIdx.y;

	if ( col < cols && row < rows ) {
		A[col * rows + row] = row;  // column major ordering
	}
}


/**
* Matrix equals to kernel checks if each element in the matrix is equal to
* the provided value. If not then it stops the block's execution.
**/
__global__ void matrix_equals_to(const double * const A, const double value, const int rows, const int cols) {
	
	const int col = blockIdx.x * blockDim.x + threadIdx.x,
			  row = blockIdx.y * blockDim.y + threadIdx.y;

	if ( col < cols && row < rows ) {
		assert(A[col * rows + row] == value); // column major ordering
	}
}


/**
* Gpu mul kernel wrapper performs the calculation of C = A**T*A
**/
void gpu_mul(const double *A, double *C, const int rows, const int cols) {
	
	// Remember column-major implementation
	int m = cols, n = cols, k = rows;	
	int lda = rows, ldb = rows, ldc = cols;
	
	// C = beta*C + alpha*A*B
	const double alpha = 1, beta = 0;
	
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	timer.Start();
	hipblasDgemm(handle, 
				HIPBLAS_OP_T,  // A**T
				HIPBLAS_OP_N,  // A normal
				m, n, k, &alpha, 
				A, lda, 
				A, ldb, 
				&beta, C, ldc);
	timer.Stop();
	// Destroy the handle
	hipblasDestroy(handle);
}


/**
* Utility function print_matrix, prints the contents of a matrix A
**/
void print_matrix(const double *A, const int rows, const int cols) {
	int i, j;
	for (i = 0; i < rows; ++i) {
		for (j = 0; j < cols; ++j) {
			//column major print
			printf("%.2lf ", A[j * rows + i]);
		}
		printf("\n");
	}
	printf("\n");
}


/**
* Utility function cleanUp, deallocates dynamic memory on host and device
**/
void cleanUp() {
	// free GPU memory
	if (d_A) hipFree(d_A);
	if (d_C) hipFree(d_C);	

	// Free CPU memory
	if (h_A) free(h_A);
	if (h_C) free(h_C);
}


/**
* Utility function gpuAssert, checks success of cuda function calls
* and exits on failure
**/
inline void gpuAssert(hipError_t code, const char *file, int line) {
   if (code != hipSuccess) {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  cleanUp();
      exit(code);
   }
}


