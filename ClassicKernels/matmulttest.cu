#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../DIEKUHDA/kuhda.h"
#include "hip/hip_runtime.h"

// Run with nvcc -O3 -lcublas ../DIEKUHDA/kuhda.c kernels.cu matmulttest.cu
#define THREADS 32

__global__ void fill_matrix(double* A, const int rows, const int cols) {
	int counter = 0;
	for (counter = 0; counter < rows*cols; ++counter) {
		A[counter] = cols;
	}
}

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, const int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

	double tmpSum = 0.0;
	int i;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
	C[ROW * N + COL] = tmpSum;
}

//extern void hello_device_wrapper(int printme);
//extern void gpu_mul_wrapper(double const * const A, double * const C, const int rows_A, const int cols_A);

//#include <hip/hip_runtime.h>
//#include "hip/hip_runtime.h"


int main() {			
	
	// Set cuda device
	gpuErrchk(hipSetDevice(0));
	unsigned long n = 1000;

	// Containers for host and device matrices:
	matrix *h_A  = kuhdaMallocMP1(n, n); // diagonal A matrix
	matrix *h_B  = kuhdaMallocMP1(n, n); // diagonal B matrix
	matrix *h_C  = kuhdaMallocMP(n, n); // empty C matrix

	matrix *d_A  = kuhdaMallocDeviceM(n, n); 
	matrix *d_B  = kuhdaMallocDeviceM(n, n);
	matrix *d_C  = kuhdaMallocDeviceM(n, n);

	// Make streams and copy data:
	hipStream_t mainstream, copystream1, copystream2;
	gpuErrchk(hipStreamCreate(&mainstream));
	gpuErrchk(hipStreamCreate(&copystream1));
	gpuErrchk(hipStreamCreate(&copystream2));

	// Allocate the timer:
    hipEvent_t mainstart, mainstop;
	float mainstreamtimer;
    gpuErrchk(hipEventCreate(&mainstart));
	gpuErrchk(hipEventCreate(&mainstop));
	gpuErrchk(hipEventRecord(mainstart, mainstream));

	// Send A and B to device 0...
	TileHostToGPU(0, n, 0, n, h_A, d_A, copystream1);
	TileHostToGPU(0, n, 0, n, h_B, d_B, copystream2);

	gpuErrchk(hipStreamSynchronize(copystream1));
	gpuErrchk(hipStreamSynchronize(copystream2));

	// Grid dimmensions for multiplication
	//grid = dim3(ceil(((float)cols_C)/block.x), ceil(((float)rows_C)/block.y));
	
	// Perform the multiplications	
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipEventRecord(mainstart, mainstream));

	//fill_matrix<<<10, 10>>>(d_C->data, d_C->r, d_C->c);
	
	int N = n;
	dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N*N > 512){
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
	}
		
	matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(d_A->data, d_B->data, d_C->data, n);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	gpuErrchk(hipEventRecord(mainstop, mainstream));
    gpuErrchk(hipEventSynchronize(mainstop));
    gpuErrchk(hipEventElapsedTime(&mainstreamtimer, mainstart, mainstop));
	printf("Multiplication on device 0 took %lf seconds\n", mainstreamtimer/1000);
	
	//gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	
	TileGPUAddToHost(0, n, 0, n, d_C, h_C, copystream1);
	gpuErrchk(hipStreamSynchronize(copystream1));

	// Test the result
	kuhdaTestM(0, n, 0, n, h_C);
	//kuhdaPrintM(h_C);
	
	// free all matrices
    printf("Cleaning up ..\n");
    gpuErrchk(hipStreamDestroy(mainstream));
	gpuErrchk(hipStreamDestroy(copystream1));
	gpuErrchk(hipStreamDestroy(copystream2));
	
    gpuErrchk(hipEventDestroy(mainstart));
	gpuErrchk(hipEventDestroy(mainstop));

	kuhdaFreeM(h_A, 'p');
	kuhdaFreeM(h_B, 'p');
	kuhdaFreeM(h_C, 'p');
	kuhdaFreeM(d_A, 'c');
	kuhdaFreeM(d_B, 'c');
	kuhdaFreeM(d_C, 'c');
	return 0;
}
