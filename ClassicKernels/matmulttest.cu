#include "hip/hip_runtime.h"
#include <stdio.h>
extern "C" {
#include "../DIEKUHDA/kuhda.h"
}

// Run with nvcc -O3 -arch=sm_70 -lcublas ../DIEKUHDA/kuhda.c kernels.cu matmulttest.cu
#define THREADS 32

// Inclusion of .cu in seperate file is necessary:
// See: https://stackoverflow.com/questions/30247592/compiling-and-linking-pure-c-and-cuda-code-warning-implicit-declaration-of-fun
#include "kernels.cuh"
#include "hip/hip_runtime.h"

//extern void hello_device_wrapper(int printme);
//extern void gpu_mul_wrapper(double const * const A, double * const C, const int rows_A, const int cols_A);

//#include <hip/hip_runtime.h>
//#include "hip/hip_runtime.h"


int main() {				
	unsigned long n = 1000, size = n * n * sizeof(double);
	unsigned long x = n/2, sizex = x * x * sizeof(double); // x * x = dimension of quarter tile

	// Containers for host and device matrices:
	matrix *h_A  = kuhdaMallocMP1(n, n); // diagonal A matrix
	matrix *h_B  = kuhdaMallocMP1(n, n); // diagonal B matrix
	matrix *h_C  = kuhdaMallocMP(n, n); // empty C matrix

	matrix *d_A  = kuhdaMallocDeviceM(n, n); 
	matrix *d_B  = kuhdaMallocDeviceM(n, n);
	matrix *d_C  = kuhdaMallocDeviceM(n, n);

	// Make streams and copy data:
	hipStream_t mainstream, copystream1, copystream2;
	gpuErrchk(hipStreamCreate(&mainstream));
	gpuErrchk(hipStreamCreate(&copystream1));
	gpuErrchk(hipStreamCreate(&copystream2));

	// Allocate the timer:
    hipEvent_t mainstart, mainstop;
	float mainstreamtimer;
    gpuErrchk(hipEventCreate(&mainstart));
	gpuErrchk(hipEventCreate(&mainstop));
	gpuErrchk(hipEventRecord(mainstart, mainstream));

	// Send A and B to device 0...
	TileHostToGPU(0, n, 0, n, h_A, d_A, copystream1);
	TileHostToGPU(0, n, 0, n, h_B, d_B, copystream2);


	int testint = 12;
	hello_device_wrapper(testint);
	
	// Set cuda device
	gpuErrchk(hipSetDevice(0));


	/*
	dim3 block(THREADS, THREADS);
	dim3 grid(ceil(((float)cols_A)/block.x), ceil(((float)rows_A)/block.y));
	*/

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Grid dimmensions for multiplication
	/*
	grid = dim3(ceil(((float)cols_C)/block.x), ceil(((float)rows_C)/block.y));
	
	// Perform the multiplications	
	gpuErrchk(hipEventRecord(mainstart, mainstream));

	gpu_mul<<<grid, block>>>(d_A, d_C, rows_A, cols_A);
	*/

	gpuErrchk(hipEventRecord(mainstop, mainstream));
    gpuErrchk(hipEventSynchronize(mainstop));
    gpuErrchk(hipEventElapsedTime(&mainstreamtimer, mainstart, mainstop));
	printf("Multiplication on device 0 took %lf seconds\n", mainstreamtimer/1000);
	
	//gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// Test the result
    //kuhdaTestM(0, n, 0, n, h_C);
	
	// free all matrices
    printf("Cleaning up ..\n");
    gpuErrchk(hipStreamDestroy(mainstream));
	gpuErrchk(hipStreamDestroy(copystream1));
	gpuErrchk(hipStreamDestroy(copystream2));
	
    gpuErrchk(hipEventDestroy(mainstart));
	gpuErrchk(hipEventDestroy(mainstop));

	kuhdaFreeM(h_A, 'p');
	kuhdaFreeM(h_B, 'p');
	kuhdaFreeM(h_C, 'p');
	kuhdaFreeM(d_A, 'c');
	kuhdaFreeM(d_B, 'c');
	kuhdaFreeM(d_C, 'c');
	return 0;
}
