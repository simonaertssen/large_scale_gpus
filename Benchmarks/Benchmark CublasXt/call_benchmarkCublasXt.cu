
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void){

    int status = 0, rep, reps = 3, iter, maxiter = 2;
    unsigned long n = 0;

    char command[sizeof(unsigned long) + 100];

    for (n = 1024; n <= 65536; n += 1024){
        for (rep = 0; rep < reps; ++rep){
            iter = 0;
            do {
                sprintf(command, "./benchmarkCublasXt %lu", n);
                // printf("command = %s", command);
                status = system(command);
                // printf("status = %d\n", status/255);
                ++iter;
            } while (status/256 != 0 and iter < maxiter);
            if (status/256 != 0 && iter == maxiter) return -1;
        }
    }
    return 0;
}