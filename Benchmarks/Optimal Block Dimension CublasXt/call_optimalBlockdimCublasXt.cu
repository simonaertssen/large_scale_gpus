
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void){

    int status = 0, rep, reps = 3, iter, maxiter = 2;
    unsigned long n = 16384, blockdim = 0;

    char command[2*sizeof(unsigned long) + 100];

    for (blockdim = 1024; blockdim <= 16384; blockdim += 1024){
        for (rep = 0; rep < reps; ++rep){
            iter = 0;
            do {
                sprintf(command, "./optimalBlockdimCublasXt %lu %lu", n, blockdim);
                // sprintf(command, "./Optimal\\ Block\\ Dimension\\ CublasXt/optimalBlockdimCublasXt %lu %lu", n, blockdim);
                // printf("command = %s", command);
                status = system(command);
                // printf("status = %d\n", status/255);
                ++iter;
            } while (status/256 != 0 && iter < maxiter);
            if (status/256 != 0 && iter == maxiter) return -1;
        }
    }
    return 0;
}