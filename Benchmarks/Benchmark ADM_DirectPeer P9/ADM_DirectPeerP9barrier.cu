#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../../DIEKUHDA/kuhda.h"
#include "omp.h"

#define NUMTHREADSBUFF 16
#define MAXSTREAMSPERD 2
#define A 0
#define B 1
#define C 2

/*
This script builds on ADM_Direct.cu, but makes use of peer to peer communication between devices.
Each device is associated with different tiles of C, and each device only computes it's own tiles of C.
All jobs are statically scheduled: one for loop over the devices, one for loop for every of the four streams on the device. 
Tiles are sent to the devices and then broadcasted between devices with a fast connection.
For the PS9, the connection between host and devices 2 and 3 is the fastest. Then only half the speed between host and device 0.
So: send to device 0 -> 1 and send to device 2 -> 3

run with
nvcc -o ADM_DirectPeerP9 -O3 -Xcompiler -fopenmp -Xcompiler -mno-float128 -lcublas ../../DIEKUHDA/kuhda.cu ADM_DirectPeerP9.cu && ADM_DirectPeerP9 8192 2048
*/

#define LOG(X,Y) fprintf(logfile, "%s, %s(%d) " #X " " #Y "\n", __TIMESTAMP__, __FILE__, __LINE__);

void TileHostToGPUBuff(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *h_matrix, matrix *d_tile, hipStream_t stream, matrix *memacc );
void TileGPUToHostBuff(unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *d_tile, matrix *h_matrix, hipStream_t stream, matrix *memacc );


int main(int argc, char* argv[]) {
    // Parallel device warmup by handle creation instead of kuhdaWarmupDevice(device);
    int device, peerdevice, devicecount = 4;

    FILE *logfile = fopen("logfile_benchmarkADM_DirectPeerP9.txt", "a");
	// freopen("logfile_benchmarkCublasXt.txt","a",stdout);
	FILE *output = fopen("results_benchmarkADM_DirectPeerP9.txt", "a");
	if (logfile == NULL || output == NULL) {
		fclose(logfile);
		fclose(output);
    return 1;
  	}
	LOG(START, SUCCES);

    omp_set_nested(true);
    hipblasHandle_t handles[devicecount]; 
    #pragma omp parallel for private(device) num_threads(devicecount)
    for (device = 0; device < devicecount; device ++){
        GPUCHECK(hipSetDevice(device));
        CUBLASCHECK(hipblasCreate(&handles[device]));
    }      

    // Set matrix size
    unsigned long n, x;
    if (argc > 1){
        n = (unsigned long)atoi(argv[1]);
        x = n/2;
    }
    if (argc > 2){
        x = (unsigned long)atoi(argv[2]);
    }    

    // Check dimensions with regards to the available memory: not when measuring block dimension performance
    kuhdaAdjustTileSizeForAvailableMemory(devicecount, n, x);
    if (x > n/2) x = n/2;

    printf("Matrix dimension = %lu, block size = %lu.. \n", n, x);
    int tileop, numtilesperdim = n/x, numtilestotal = numtilesperdim*numtilesperdim, numtilesperdev = numtilestotal/devicecount;
    int streamop, numtilesperstream = numtilesperdev/MAXSTREAMSPERD;
    numtilesperstream = numtilesperstream < 1 ? 1 : numtilesperstream;

    // Containers for host and device matrices
    unsigned long m = n, k = n;    
	matrix *h_A = kuhdaMallocMdiag(n, n); // matrix A as a diagonal matrix
    matrix *h_B = kuhdaMallocM(n, n); // matrix B to be filled with specific values for specific testing
    matrix *h_C = kuhdaMallocM(n, n);     // matrix C will contain results: same values at each spot as in b
    unsigned long i, j;
    #pragma omp parallel for private(i,j) num_threads(NUMTHREADSBUFF)
	for (i = 0; i < h_B->r; ++i){
		for (j = 0; j < h_B->c; ++j){
            h_B->data[i*h_B->c + j] = (i + j) * 0.1 + i;
        }
    }

    // Counters for streams: number of streams is number of operations per device, but adjust for less streams if large tiles
    int stream, numstreamsperdevice = numtilesperdev > MAXSTREAMSPERD ? MAXSTREAMSPERD : numtilesperdev;
    // printf("numtilestotal = %d, numtilesperdev = %d, numtilesperstream = %d, numstreamsperdevice = %d\n", numtilestotal, numtilesperdev, numtilesperstream, numstreamsperdevice);
    
    int abc, ABC = 3; 
    matrix *d_All[devicecount][ABC][numstreamsperdevice];   // matrix tiles on each device per stream

    // Cuda dependencies
    hipStream_t d_streams[devicecount][numstreamsperdevice];
    hipStream_t p_streams[devicecount][numstreamsperdevice];
    hipEvent_t streamReady[devicecount][numstreamsperdevice];
    matrix *membuffs[devicecount][numstreamsperdevice][2];

    MatMulTimer timer;

    // Parallel device memory and dependency allocation
    // printf("Allocating tiles A, B and C on %d devices..\n", devicecount);
    #pragma omp parallel for private(device, abc, stream) num_threads(devicecount)
    // Creat all dependencies:
    for (device = 0; device < devicecount; ++device){
        GPUCHECK(hipSetDevice(device));
        // Make device able to send and receive data from peers:
		for (peerdevice = 0; peerdevice < devicecount; ++peerdevice){
			if (peerdevice == device) continue;
            // GPUCHECK(hipDeviceEnablePeerAccess(peerdevice, 0));
        }

        #pragma omp parallel for private(stream, abc) num_threads(numstreamsperdevice)
        for (stream = 0; stream < numstreamsperdevice; ++stream){
            for (abc = 0; abc < ABC; ++abc){
                d_All[device][abc][stream] = kuhdaMallocDeviceM(x, x);
            }
            GPUCHECK(hipStreamCreate(&d_streams[device][stream]));
            GPUCHECK(hipStreamCreate(&p_streams[device][stream]));
            GPUCHECK(hipEventCreateWithFlags(&streamReady[device][stream], hipEventBlockingSync));
            // GPUCHECK(hipStreamCreateWithFlags(&d_streams[device][stream], hipStreamNonBlocking));
            membuffs[device][stream][0] = kuhdaMallocMP(x, x);
            membuffs[device][stream][1] = kuhdaMallocMP(x, x);
        }
    }

    // Main loop counters:
    int Arow, Acol, Brow, Bcol, Crow, Ccol, tileindex, printonce = 0; 

    printf("Computation start..\n");
    timer.Start();

    // Parallel device multiplication loop
    #pragma omp parallel num_threads(devicecount)
    {
    #pragma omp for 
    for (device = 0; device < devicecount; device++){
        GPUCHECK(hipSetDevice(device));

        // Loop over streams per device
        #pragma omp parallel private(stream, streamop, tileindex, tileop, Arow, Acol, Brow, Bcol, Crow, Ccol) num_threads(numstreamsperdevice) 
        {
        #pragma omp for 
        for (stream = 0; stream < numstreamsperdevice; ++stream){
            // Loop over all operations on C per stream
            // #pragma omp parallel for private(tileindex, tileop, Arow, Acol, Brow, Bcol, Crow, Ccol) num_threads(numtilesperstream)
            for (streamop = 0; streamop < numtilesperstream; ++streamop){
                // Register indices of C tiles
                // tileindex = (device*numstreamsperdevice + stream)*numtilesperstream + streamop; 
                tileindex = device + (stream*numtilesperstream + streamop)*devicecount; //(streamop*devicecount + device)*numstreamsperdevice + stream; 
                // printf("device %d, stream %d, tileindex %d: test = %d\n", device, stream, tileindex, (tileindex+1) % numtilesperdim);
                Crow = tileindex/numtilesperdim; Ccol = tileindex%numtilesperdim;

                // Set contents of C to zero for use as an accumulator:
                GPUCHECK(hipMemsetAsync(d_All[device][C][stream]->data, 0, x*x*sizeof(double), d_streams[device][stream]));
                // printf("Dev %d, stream %d: tileindex = %d, (%d,%d)\n", device, stream, tileindex, Crow, Ccol);

                printonce = 1;
                // Loop over all tiles of A and B to copy: Arow = Crow and Bcol = Ccol
                // #pragma omp parallel for private(tileop, Arow, Acol, Brow, Bcol) num_threads(numtilesperdim)
                for (tileop = 0; tileop < numtilesperdim; ++tileop){
                    Arow = Crow;   Acol = tileop;
                    Brow = tileop; Bcol = Ccol;

                    // Copy the A-tile between devices, using the same stream number of the devices.
                    // If the following C tile still has the same row of A, send the tile between devices, otherwise not.
                    // Check first if the current tile is not the last in the row (+1 for 0-based indexing).
                    // Then check whether the next tile is on the same row, which should be the same multiple of numtilesperdim.
                    // printf("%d ?= %d\n", (tileindex)/numtilesperdim, (tileindex+1)/numtilesperdim);
                    if (device == 0 || device == 2){
                        TileHostToGPUBuff(Arow*x, (Arow+1)*x, Acol*x, (Acol+1)*x, h_A, d_All[device][A][stream], d_streams[device][stream], membuffs[device][stream][0]); // Tile A
                        // Now send from 0->1 and from 2->3
                        if (((tileindex+1) % numtilesperdim != 0) && Crow == (tileindex+1)/numtilesperdim){
                            GPUCHECK(hipStreamSynchronize(d_streams[device][stream]));
                            // inputs are: (void* dst, int  dstDevice, const void* src, int  srcDevice, size_t count, hipStream_t stream = 0)
                            GPUCHECK(hipMemcpyPeerAsync(d_All[device+1][A][stream]->data, device+1, d_All[device][A][stream]->data, device, x*x*sizeof(double), d_streams[device][stream]));
                            // GPUCHECK(hipMemcpyAsync(d_All[device+1][A][stream]->data, d_All[device][A][stream]->data, x*x*sizeof(double), hipMemcpyDeviceToDevice));
                            GPUCHECK(hipStreamSynchronize(d_streams[device][stream]));
                            if (printonce) printf("%d: Sending from %d to %d\n", device, tileindex, tileindex+1);
                            GPUCHECK(hipEventRecord(streamReady[device][stream], d_streams[device][stream]));
                        } else {
                            if (printonce) printf("%d: No sending possible from %d to %d\n", device, tileindex, tileindex+1);
                        }
                    }
                    #pragma omp barrier
                    if (device == 1 || device == 3){
                        if (Crow == (tileindex-1)/numtilesperdim){
                            // GPUCHECK(hipEventSynchronize(streamReady[device-1][stream]));
                            // printf("Status of the event = %d", hipEventQuery(streamReady[device-1][stream])
                            
                            GPUCHECK(hipStreamWaitEvent(d_streams[device][stream], streamReady[device-1][stream], 0));
                            // GPUCHECK(hipStreamSynchronize(d_streams[device-1][stream]));
                            if (printonce) printf("%d: Receiving from %d on %d\n", device, tileindex-1, tileindex);
                        } else {
                            if (printonce) printf("%d: No receiving possible from %d on %d\n", device, tileindex-1, tileindex);
                            TileHostToGPUBuff(Arow*x, (Arow+1)*x, Acol*x, (Acol+1)*x, h_A, d_All[device][A][stream], d_streams[device][stream], membuffs[device][stream][0]); // Tile A
                        }
                    }
                    
                    // TileHostToGPUBuff(Arow*x, (Arow+1)*x, Acol*x, (Acol+1)*x, h_A, d_All[device][A][stream], d_streams[device][stream], membuffs[device][stream][0]); // Tile A
                    TileHostToGPUBuff(Brow*x, (Brow+1)*x, Bcol*x, (Bcol+1)*x, h_B, d_All[device][B][stream], d_streams[device][stream], membuffs[device][stream][1]); // Tile B
                    
                    GPUCHECK(hipStreamSynchronize(d_streams[device][stream]));
                    kuhdammson(d_All[device][A][stream], d_All[device][B][stream], d_All[device][C][stream], d_streams[device][stream], handles[device]);
                }

                TileGPUToHostBuff(Crow*x, (Crow+1)*x, Ccol*x, (Ccol+1)*x, d_All[device][C][stream], h_C, d_streams[device][stream], membuffs[device][stream][0]);
                printf("sending tile %d back\n", tileindex);

            }
        }
        }
        hipDeviceSynchronize();
    }
}
    
    timer.Stop();
    double timingResult = timer.GFLOPS_DGEMM(m, n, k);
    printf("GFLOPS = %.0lf..\n", timingResult);

    // Test the result for mistakes
    printf("Checking results. ");
    double abserror = 0.0, totalerror = 0.0;
    #pragma omp parallel for private(i,j) num_threads(NUMTHREADSBUFF) reduction(+:totalerror)
	for (i = 0; i < h_B->r; ++i){
		for (j = 0; j < h_B->c; ++j){
            abserror = fabs(h_B->data[i*h_B->c + j] - h_C->data[i*h_C->c + j]);
            totalerror += abserror;
            if (abserror > 10e-6) {
                // printf("Failure: B[%d] = %1.4e != C[%d] = %1.4e\n", i*h_B->c + j, h_B->data[i*h_B->c + j], i*h_C->c + j, h_C->data[i*h_C->c + j]);
                break;
            }
        }
    }
    if (totalerror < 10e-6) printf("Succes. ");
    printf("Total error of %6.2e..\n", totalerror);
    fprintf(output, "%zu, %d, %.1lf\n", n, x, timingResult);

    // Free all dependencies
    // printf("Cleaning up..\n");
    GPUCHECK(hipSetDevice(0));

	kuhdaFreeM(h_A, 'k');
	kuhdaFreeM(h_B, 'k');
    kuhdaFreeM(h_C, 'k');

    timer.Release();
 
    #pragma omp parallel for private(device, abc, stream) num_threads(devicecount) 
    for (device = 0; device < devicecount; device++){
        GPUCHECK(hipSetDevice(device));
        CUBLASCHECK(hipblasDestroy(handles[device]));

        #pragma omp parallel for private(stream, abc) num_threads(numstreamsperdevice)
        for (stream = 0; stream < numstreamsperdevice; ++stream){
            for (abc = 0; abc < ABC; ++abc){
                kuhdaFreeM(d_All[device][abc][stream], 'c');
            }
            GPUCHECK(hipStreamDestroy(d_streams[device][stream]));
            GPUCHECK(hipStreamDestroy(p_streams[device][stream]));
            kuhdaFreeM(membuffs[device][stream][0], 'p');
            kuhdaFreeM(membuffs[device][stream][1], 'p');
            GPUCHECK(hipEventDestroy(streamReady[device][stream]));
        }
        // Takes NO arguments
        GPUCHECK(hipDeviceReset());
    }

    LOG(STOP, SUCCES);
	fclose(logfile);
	fclose(output);
	return 0;
}


void TileHostToGPUBuff(	unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *h_matrix, matrix *d_tile, hipStream_t stream, matrix *memacc ) {	
    if (h_matrix == NULL || d_tile == NULL) INPUT_NULL_ERR;
    if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
    if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
    if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
    if (stream == NULL) INPUT_NULL_ERR;

    unsigned long i, j; //cols = colstop - colstart, rows = rowstop - rowstart, i, j;

    #pragma omp parallel for private(i,j) num_threads(NUMTHREADSBUFF) collapse(2)
    for (i=rowstart; i<rowstop; ++i){
        for (j=colstart; j<colstop; ++j){
            memacc->data[(i - rowstart) * memacc->c + (j - colstart)] = h_matrix->data[i * h_matrix->c + j];
        }
    }
    
    // GPUCHECK(hipMemcpyAsync((void*)&d_tile->data[0], (void*)&memacc->data[0], rows*cols*sizeof(double), hipMemcpyHostToDevice, stream));
    GPUCHECK(hipMemcpy2DAsync((void*)&d_tile->data[0], memacc->c*sizeof(double), (const void*)&memacc->data[0], memacc->c*sizeof(double), memacc->c*sizeof(double), memacc->r, hipMemcpyHostToDevice, stream));

}

void TileGPUToHostBuff( unsigned long rowstart, unsigned long rowstop, unsigned long colstart, unsigned long colstop, 
    matrix *d_tile, matrix *h_matrix, hipStream_t stream, matrix *memacc ){
    if (h_matrix == NULL || d_tile == NULL) INPUT_NULL_ERR;
    if (rowstart > rowstop) INPUT_ILL_ERR_LU(rowstop);
    if (colstart > colstop)	INPUT_ILL_ERR_LU(colstop);
    if (h_matrix->r <= 0 || h_matrix->c <= 0 || d_tile->r <= 0 || d_tile->c <= 0) INPUT_ILL_ERR_LU(h_matrix->r);
    if (stream == NULL) INPUT_NULL_ERR;

    unsigned long i, j; //cols = colstop - colstart, rows = rowstop - rowstart, i, j;
    // GPUCHECK(hipMemcpyAsync((void*)&memacc->data[0], (void*)&d_tile->data[0], rows*cols*sizeof(double), hipMemcpyDeviceToHost, stream));
    GPUCHECK(hipMemcpy2DAsync((void*)&memacc->data[0], memacc->c*sizeof(double), (const void*)&d_tile->data[0], d_tile->c*sizeof(double), d_tile->c*sizeof(double), d_tile->r, hipMemcpyDeviceToHost, stream));
    GPUCHECK(hipStreamSynchronize(stream));

    #pragma omp parallel for private(i,j) num_threads(NUMTHREADSBUFF) collapse(2)
    for (i = rowstart; i < rowstop; ++i){
        for (j = colstart; j < colstop; ++j){
            h_matrix->data[i * h_matrix->c + j] = memacc->data[(i - rowstart) * memacc->c + (j - colstart)];
        }
    }
}
